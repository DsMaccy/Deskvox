#include "hip/hip_runtime.h"
// Virvo - Virtual Reality Volume Rendering
// Copyright (C) 2010 University of Cologne
// Contact: Jurgen P. Schulze, jschulze@ucsd.edu
//
// This file is part of Virvo.
//
// Virvo is free software; you can redistribute it and/or
// modify it under the terms of the GNU Lesser General Public
// License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
//
// This library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public
// License along with this library (see license.txt); if not, write to the 
// Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA

#include <iostream>
using std::cerr;
using std::endl;

#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include "vvdebugmsg.h"
#include "vvvecmath.h"
#include "vvstopwatch.h"
#include "vvcuda.h"
#include "vvcudapar.h"

const int MAX_SLICES = 512;

__constant__ int   c_vox[5];
__constant__ float2 c_start[MAX_SLICES];
static float2 h_start[MAX_SLICES];

const int MaxCompositeSlices = 1024;

//#define SHMCLASS
//#define NOOP
//#define NOLOAD
//#define NODISPLAY
//#define SHMLOAD
#define VOL3TEX3D
//#define PITCHED
#define FLOATDATA
//#define CONSTLOAD
//#define THREADPERVOXEL
//#define CONSTDATA
//#define FLOATIMG
#define EARLYRAYTERM

const int Repetitions = 1;

#ifdef VOL3TEX3D
#undef PITCHED
#undef CONSTLOAD
#undef CONSTDATA
#undef NOLOAD
#undef SHMLOAD
#undef SHMCLASS
#endif

#ifdef CONSTDATA
#undef SHMLOAD
#endif

#ifdef CONSTLOAD
#undef NOLOAD
#endif

#ifdef NOLOAD
#define SHMLOAD
#endif

#ifdef SHMCLASS
#define SHMLOAD
texture<uchar4, 1, hipReadModeElementType> tex_tf;
#else
texture<uchar4, 1, hipReadModeNormalizedFloat> tex_tf;
#endif

#ifdef VOL3TEX3D
#ifdef FLOATDATA
texture<float, 3, hipReadModeElementType> tex_raw;
#else
texture<uchar, 3, hipReadModeNormalizedFloat> tex_raw;
#endif
#endif

#ifdef FLOATDATA
typedef float Scalar;
#else
typedef uchar Scalar;
#endif

typedef void (*CompositionFunction)(
      uchar4 * __restrict__ img, int width, int height,
#ifdef PITCHED
      const hipPitchedPtr pvoxels,
#else
      const Scalar * __restrict__ voxels,
#endif
      int firstSlice, int lastSlice,
      int from, int to);

//----------------------------------------------------------------------------
// device code (CUDA)
//----------------------------------------------------------------------------

__global__ void clearImage(uchar4 * __restrict__ img, int width, int height,
      int from, int to)
{
    const int line = blockIdx.x+from;
    if (line >= to)
        return;

    for (int ix=threadIdx.x; ix<width; ix+=blockDim.x)
    {
        uchar4 *dest = img + line*width+ix;
        *dest = make_uchar4(0, 0, 0, 0);
    }
}

template<typename Scalar, int BPV, int sliceStep, int principal>
__global__ void compositeSlicesNearest(
      uchar4 * __restrict__ img, int width, int height,
#ifdef PITCHED
      const hipPitchedPtr pvoxels,
#else
      const Scalar * __restrict__ voxels,
#endif
      int firstSlice, int lastSlice,
      int from, int to)
{
#ifdef PITCHED
    const Scalar *voxels = (Scalar *)pvoxels.ptr;
    const int pitch = pvoxels.pitch;
#else
    const int pitch = c_vox[principal] * BPV * sizeof(Scalar);
#endif
    const int line = blockIdx.x+from;
    if (line >= to)
        return;

    // initialise intermediate image line
    extern __shared__ char smem[];
#ifdef FLOATIMG
    float4 *imgLine = (float4 *)smem;
    const int pixsize = sizeof(float4);
#else
    uchar4 *imgLine = (uchar4 *)smem;
    const int pixsize = sizeof(uchar4);
#endif
#ifdef SHMLOAD
#ifdef SHMCLASS
    uchar4 *voxel = (uchar4 *)(smem+width*pixsize);
#else
    Scalar *voxel = (Scalar *)(smem+width*pixsize);
#endif
#endif

    for (int ix=threadIdx.x; ix<width; ix+=blockDim.x)
    {
#ifdef FLOATIMG
        imgLine[ix] = make_float4(0.f,0.f,0.f,1.f);
#else
        imgLine[ix] = make_uchar4(0,0,0,255);
#endif
    }

    for(int i=0; i<Repetitions; ++i)
    {

    // composite slices for this image line
    for (int slice=firstSlice; slice!=lastSlice; slice += sliceStep)
    {
#ifdef CONSTLOAD
        const Scalar *voxLine = (Scalar *)(((uchar *)voxels) + pitch * c_vox[principal+1]);
#endif
#ifdef NOOP
        const int iPosY = line;
#else
        // compute upper left image corner
        const int iPosY = float2int(c_start[slice].y+0.5f);

        if(line < iPosY)
            continue;
        if(line >= iPosY+c_vox[principal+1])
            continue;

        const int iPosX = float2int(c_start[slice].x+0.5f);
#endif

        // the voxel row of the current slice corresponding to this image line
#ifndef NOLOAD
#ifndef CONSTLOAD
        const Scalar *voxLine = (Scalar *)(((uchar *)voxels) + pitch * ((slice+1)*c_vox[principal+1] + (iPosY-line-1)));
#endif

#ifdef SHMLOAD
        for (int ix=threadIdx.x; ix<c_vox[principal+0]; ix+=blockDim.x)
        {
#ifdef SHMCLASS
#ifdef FLOATDATA
            voxel[ix] = tex1Dfetch(tex_tf, voxLine[ix]*255.f);
#else
            voxel[ix] = tex1Dfetch(tex_tf, voxLine[ix]);
#endif
#else
            voxel[ix] = voxLine[ix];
#endif
        }
        __syncthreads();
#endif
#endif

#ifndef NOOP
        // Traverse intermediate image pixels which correspond to the current slice.
        // 1 is subtracted from each loop counter to remain inside of the volume boundaries:
#ifdef THREADPERVOXEL
        for (int ix=threadIdx.x; ix<c_vox[principal+0]; ix+=blockDim.x)
#else
        for (int ix=threadIdx.x; ix<c_vox[principal+0]+iPosX; ix+=blockDim.x)
#endif
        {
#ifndef THREADPERVOXEL
            if(ix<iPosX)
                continue;
#endif
#ifdef THREADPERVOXEL
            const int vidx = ix;
            const int iidx = ix + iPosX;
#else
            const int vidx = ix - iPosX;
            const int iidx = ix;
#endif

            // pointer to destination pixel
#ifdef FLOATIMG
            float4 *pix = imgLine + iidx;
            float4 d = *pix;
#ifdef EARLYRAYTERM
            if(d.w < 0.001f)
                continue;
#endif
#else
            uchar4 *pix = imgLine + iidx;
            uchar4 d = *pix;
#ifdef EARLYRAYTERM
            if(d.w < 1)
                continue;
#endif
#endif

#ifdef VOL3TEX3D
            const float v = tex3D(tex_raw, float(vidx), float(c_vox[principal+1]+iPosY-line-1), float(slice));
            const float4 c = tex1Dfetch(tex_tf, v*255.f);
#else
#ifdef CONSTDATA
            const float4 c = tex1Dfetch(tex_tf, ix);
#else
#ifdef SHMCLASS
            const uchar4 v = *(voxel + BPV * vidx);
            const float4 c = make_float4(v.x/255.f, v.y/255.f, v.z/255.f, v.w/255.f);
#else
            // fetch scalar voxel value
#ifdef SHMLOAD
            const Scalar *v = voxel + BPV * vidx;
#else
            const Scalar *v = voxLine + BPV * vidx;
#endif
            // apply transfer function
#ifdef FLOATDATA
            const float4 c = tex1Dfetch(tex_tf, *v*255.f);
#else
            const float4 c = tex1Dfetch(tex_tf, *v);
#endif
#endif
#endif
#endif

            // blend
            const float w = d.w*c.w;
            d.x += w*c.x;
            d.y += w*c.y;
            d.z += w*c.z;
            d.w -= w;

            // store into shmem
            *pix = d;
#ifdef THREADPERVOXEL
            __syncthreads();
#endif
        }
#endif
    }
    }

#ifndef NODISPLAY
    // copy line to intermediate image
    for (int ix=threadIdx.x; ix<width; ix+=blockDim.x)
    {
        uchar4 *dest = img + line*width+ix;
        uchar4 c = *dest;
#ifdef FLOATIMG
        *dest = make_uchar4(imgLine[ix].x * 255.f + c.x*imgLine[ix].w,
                imgLine[ix].y * 255.f + c.y*imgLine[ix].w,
                imgLine[ix].z * 255.f + c.z*imgLine[ix].w,
                (1.f-imgLine[ix].w) * (255-c.w) + c.w);
#else
        const float w= imgLine[ix].w/255.f;
        imgLine[ix].w = 255 - imgLine[ix].w;
        *dest = make_uchar4(imgLine[ix].x + c.x*w,
                imgLine[ix].y + c.y*w,
                imgLine[ix].z + c.z*w,
                (1.f-w) * (255-c.w) + c.w);
#endif
    }
#endif
}


//----------------------------------------------------------------------------
// host code
//----------------------------------------------------------------------------


//----------------------------------------------------------------------------
/** Constructor.
  @param vd volume description of volume to display
  @see vvRenderer
*/
vvCudaPar::vvCudaPar(vvVolDesc* vd, vvRenderState rs) : vvSoftPar(vd, rs)
{
   vvDebugMsg::msg(1, "vvCudaPar::vvCudaPar()");

   rendererType = CUDAPAR;
   mappedImage = false;
   if (warpMode==TEXTURE)
   {
       // we need a power-of-2 image size for glTexImage2D
       int imgSize = vvToolshed::getTextureSize(2 * ts_max(vd->vox[0], vd->vox[1], vd->vox[2]));

#ifndef NODISPLAY
       if (vvCuda::initGlInterop())
       {
           vvDebugMsg::msg(1, "using CUDA/GL interop");
           // avoid image copy from GPU to CPU and back
           setWarpMode(CUDATEXTURE);
           intImg->setSize(imgSize, imgSize, NULL, true);
       }
       else
       {
         vvDebugMsg::msg(1, "can't use CUDA/GL interop");
         intImg->setSize(imgSize, imgSize);
       }
#endif
   }

   wViewDir.set(0.0f, 0.0f, 1.0f);

#ifdef FLOATDATA
   for(int i=0; i<3; ++i)
   {
       size_t vox = vd->vox[0]*vd->vox[1]*vd->vox[2];
       fraw[i] = new float[vox];
       for(size_t j=0; j<vox; ++j)
       {
           fraw[i][j] = raw[i][j] / 255.f;
       }
   }
#endif

   bool ok = true;
#if defined(PITCHED) || defined(VOL3TEX3D)
   for (int i=0; i<3; ++i)
   {
#ifdef PITCHED
       hipExtent extent = make_hipExtent(vd->vox[(i+1)%3]*sizeof(Scalar), vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
       if(!vvCuda::checkError(&ok, hipMalloc3D(&d_voxptr[i], extent), "hipMalloc3D vox"))
           break;
#else
       hipExtent extent = make_hipExtent(vd->vox[(i+1)%3], vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
       hipChannelFormatDesc desc = hipCreateChannelDesc<Scalar>();
       if(!vvCuda::checkError(&ok, hipMalloc3DArray(&d_voxarr[i], &desc, extent, 0), "hipMalloc3DArray vox"))
           break;
#endif
       hipMemcpy3DParms parms = {0};
#ifdef FLOATDATA
       parms.srcPtr = make_hipPitchedPtr(fraw[i], sizeof(Scalar)*vd->vox[(i+1)%3], vd->vox[(i+1)%3], vd->vox[(i+2)%3]);
#else
       parms.srcPtr = make_hipPitchedPtr(raw[i], sizeof(Scalar)*vd->vox[(i+1)%3], vd->vox[(i+1)%3], vd->vox[(i+2)%3]);
#endif

#ifdef PITCHED
       parms.dstPtr = d_voxptr[i];
       parms.extent = make_hipExtent(vd->vox[(i+1)%3]*sizeof(Scalar), vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
#else
       parms.dstArray = d_voxarr[i];
       parms.extent = make_hipExtent(vd->vox[(i+1)%3], vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
#endif
       parms.kind = hipMemcpyHostToDevice;
       if(!vvCuda::checkError(&ok, hipMemcpy3D(&parms), "hipMemcpy3D vox"))
           break;
   }
#else
   // alloc memory for voxel arrays (for each principal viewing direction)
   vvCuda::checkError(&ok, hipMalloc(&d_voxels, sizeof(Scalar)*vd->vox[0]*vd->vox[1]*vd->vox[2]*3), "hipMalloc vox");
   for (int i=0; i<3; ++i)
   {
#ifdef FLOATDATA
       if (!vvCuda::checkError(&ok, hipMemcpy(d_voxels+i*sizeof(Scalar)*vd->vox[0]*vd->vox[1]*vd->vox[2],
                   fraw[i], sizeof(Scalar)*vd->getFrameBytes(), hipMemcpyHostToDevice), "hipMemcpy vox"))
#else
       if (!vvCuda::checkError(&ok, hipMemcpy(d_voxels+i*sizeof(Scalar)*vd->vox[0]*vd->vox[1]*vd->vox[2],
                   raw[i], vd->getFrameBytes(), hipMemcpyHostToDevice), "hipMemcpy vox"))
#endif
          break;
   }
#endif

   // transfer function is stored as a texture
   vvCuda::checkError(&ok, hipMalloc(&d_tf, 4096*4), "hipMalloc tf");
   vvCuda::checkError(&ok, hipBindTexture(NULL, tex_tf, d_tf, 4096), "bind tf tex");

#ifndef NODISPLAY
   // allocate output image (intermediate image)
   if (warpMode==CUDATEXTURE)
   {
       vvCuda::checkError(&ok, hipGraphicsGLRegisterBuffer(&intImgRes, intImg->getPboName(), cudaGraphicsMapFlagsWriteDiscard), "map PBO to CUDA");
   }
   else if (mappedImage)
   {
       vvCuda::checkError(&ok, hipHostAlloc(&h_img, intImg->width*intImg->height*vvSoftImg::PIXEL_SIZE, hipHostMallocMapped), "img alloc");;
       intImg->setSize(intImg->width, intImg->height, h_img, false);
       vvCuda::checkError(&ok, hipHostGetDevicePointer(&d_img, h_img, 0), "get dev ptr img");
   }
   else
#endif
   {
       vvCuda::checkError(&ok, hipMalloc(&d_img, intImg->width*intImg->height*vvSoftImg::PIXEL_SIZE), "hipMalloc img");
   }

   // copy volume size (in voxels)
   int h_vox[5];
   for (int i=0; i<5; ++i)
       h_vox[i] = vd->vox[(i+1)%3];
   vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_vox), h_vox, sizeof(int)*5), "hipMemcpy vox");

   updateTransferFunction();
}


//----------------------------------------------------------------------------
/// Destructor.
vvCudaPar::~vvCudaPar()
{
   vvDebugMsg::msg(1, "vvCudaPar::~vvCudaPar()");

#ifdef FLOATDATA
   for(int i=0; i<3; ++i)
     delete[] fraw[i];
#endif

#ifndef NODISPLAY
   if (warpMode==CUDATEXTURE)
      hipGraphicsUnregisterResource(intImgRes);
   else if (mappedImage)
       hipHostFree(h_img);
   else
#endif
       hipFree(d_img);

   hipUnbindTexture(tex_tf);

   hipFree(d_tf);
#ifdef VOL3TEX3D
   for(int i=0; i<3; ++i)
     hipFree(d_voxarr[i]);
#else
#ifdef PITCHED
   for(int i=0; i<3; ++i)
       hipFree(d_voxptr[i].ptr);
#else
   hipFree(d_voxels);
#endif
#endif
}

void vvCudaPar::updateTransferFunction()
{
   vvDebugMsg::msg(2, "vvCudaPar::updateTransferFunction()");

   vvSoftPar::updateTransferFunction();
   vvCuda::checkError(NULL, hipMemcpy(d_tf, rgbaConv, sizeof(rgbaConv), hipMemcpyHostToDevice), "hipMemcpy tf");
}

template<int principal, int sliceStep>
CompositionFunction selectComposition()
{
   return compositeSlicesNearest<Scalar, 1, sliceStep, principal>;
}

template<int principal>
CompositionFunction selectCompositionWithSliceStep(int sliceStep)
{
    switch(sliceStep)
    {
        case 1:
            return selectComposition<principal,1>();
        case -1:
            return selectComposition<principal,-1>();
        default:
            assert("slice step out of range" == NULL);
    }

    return NULL;
}

CompositionFunction selectCompositionWithPrincipalAndSliceStep(int principal, int sliceStep)
{
    switch(principal)
    {
        case 0:
            return selectCompositionWithSliceStep<0>(sliceStep);
        case 1:
            return selectCompositionWithSliceStep<1>(sliceStep);
        case 2:
            return selectCompositionWithSliceStep<2>(sliceStep);
        default:
            assert("principal axis out of range" == NULL);

    }

    return NULL;
}


//----------------------------------------------------------------------------
/** Composite the volume slices to the intermediate image.
  The function prepareRendering() must be called before this method.
  The shear transformation matrices have to be computed before calling this method.
  The volume slices are processed from front to back.
  @param from,to optional arguments to define first and last intermediate image line to render.
                 if not passed, the entire intermediate image will be rendered
*/
void vvCudaPar::compositeVolume(int from, int to)
{
   vvDebugMsg::msg(3, "vvCudaPar::compositeVolume(): ", from, to);

   // If stacking==true then draw front to back, else draw back to front:
   int firstSlice = (stacking) ? 0 : (len[2]-1);  // first slice to process
   int lastSlice  = (stacking) ? (len[2]-1) : 0;  // last slice to process
   int sliceStep  = (stacking) ? 1 : -1;          // step size to get to next slice

   earlyRayTermination = 0;

   if (from == -1)
       from = 0;
   if (to == -1)
       to = intImg->height;

   // compute data for determining upper left image corner of each slice and copy it to device
   vvVector4 start, inc;
   findSlicePosition(firstSlice, &start, NULL);
   findSlicePosition(firstSlice+sliceStep, &inc, NULL);
   inc.sub(&start);
   vvVector4 cur = start;
   for(int slice=firstSlice; slice != lastSlice; slice += sliceStep)
   {
       h_start[slice].x = cur.e[0] / cur.e[3];
       h_start[slice].y = cur.e[1] / cur.e[3];
       cur.add(&inc);
   }

   bool ok = true;
   vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_start), h_start, sizeof(h_start)), "hipMemcpy start");

#ifdef VOL3TEX3D
   tex_raw.normalized = false;
   tex_raw.filterMode = hipFilterModePoint;
   tex_raw.addressMode[0] = hipAddressModeClamp;
   tex_raw.addressMode[1] = hipAddressModeClamp;
   tex_raw.addressMode[2] = hipAddressModeClamp;
   hipChannelFormatDesc desc = hipCreateChannelDesc<Scalar>();
   hipBindTextureToArray(tex_raw, d_voxarr[principal], desc);
#endif

#ifndef NODISPLAY
   // prepare intermediate image
   if (warpMode==CUDATEXTURE)
   {
       vvCuda::checkError(&ok, hipGraphicsMapResources(1, &intImgRes, NULL), "map CUDA resource");
       size_t size;
       vvCuda::checkError(&ok, hipGraphicsResourceGetMappedPointer((void**)&d_img, &size, intImgRes), "get PBO mapping");
       assert(size == intImg->width*intImg->height*vvSoftImg::PIXEL_SIZE);
   }
   else
   {
       intImg->clear();
   }
#endif

#ifdef FLOATIMG
   int shmsize = intImg->width*sizeof(float4);
#else
   int shmsize = intImg->width*vvSoftImg::PIXEL_SIZE;
#endif
#ifdef SHMLOAD
   shmsize += vd->vox[principal]*vd->getBPV()*sizeof(Scalar);
#endif

   clearImage <<<to-from, 128, shmsize>>>(d_img, intImg->width, intImg->height, from, to);

   CompositionFunction compose = selectCompositionWithPrincipalAndSliceStep(principal, sliceStep);
   // do the computation on the device
   for(int i=lastSlice; i*sliceStep>firstSlice*sliceStep; i-=sliceStep*MaxCompositeSlices)
   {
       hipDeviceSynchronize();
#ifdef PITCHED
       compose <<<to-from, 128, shmsize>>>(
               d_img, intImg->width, intImg->height,
               d_voxptr[principal],
               sliceStep*max(sliceStep*i-MaxCompositeSlices, sliceStep*firstSlice), i,
               from, to);
#else
       compose <<<to-from, 128, shmsize>>>(
               d_img, intImg->width, intImg->height,
               (Scalar *)(d_voxels+sizeof(Scalar)*vd->getBPV()*principal*(vd->vox[0]*vd->vox[1]*vd->vox[2])),
               sliceStep*max(sliceStep*i-MaxCompositeSlices, sliceStep*firstSlice), i,
               from, to);
#endif
   }

#ifdef VOL3TEX3D
   hipUnbindTexture(tex_raw);
#endif

   // copy back or unmap for using as PBO
   ok = vvCuda::checkError(&ok, hipGetLastError(), "start kernel");
#ifndef NODISPLAY
   if (warpMode==CUDATEXTURE)
   {
       vvCuda::checkError(&ok, hipGraphicsUnmapResources(1, &intImgRes, NULL), "unmap CUDA resource");
   }
   else if (mappedImage)
   {
       hipDeviceSynchronize();
   }
   else
   {
       hipMemcpy(intImg->data, d_img, intImg->width*intImg->height*vvSoftImg::PIXEL_SIZE, hipMemcpyDeviceToHost);
       ok = vvCuda::checkError(&ok, hipGetLastError(), "cpy to host");
   }
#endif
}
//============================================================================
// End of File
//============================================================================
