#include "hip/hip_runtime.h"
// Virvo - Virtual Reality Volume Rendering
// Copyright (C) 2010 University of Cologne
// Contact: Jurgen P. Schulze, jschulze@ucsd.edu
//
// This file is part of Virvo.
//
// Virvo is free software; you can redistribute it and/or
// modify it under the terms of the GNU Lesser General Public
// License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
//
// This library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public
// License along with this library (see license.txt); if not, write to the 
// Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA

#include <iostream>
using std::cerr;
using std::endl;

#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include "vvdebugmsg.h"
#include "vvvecmath.h"
#include "vvstopwatch.h"
#include "vvcuda.h"
#include "vvcudapar.h"

const int MAX_SLICES = 1600;

__constant__ int   c_vox[5];
__constant__ int2 c_start[MAX_SLICES];
static int2 h_start[MAX_SLICES];
__constant__ int2 c_stop[MAX_SLICES];
static int2 h_stop[MAX_SLICES];
__constant__ float2 c_tcStart[MAX_SLICES];
static float2 h_tcStart[MAX_SLICES];
__constant__ float2 c_tcStep[MAX_SLICES];
static float2 h_tcStep[MAX_SLICES];
__constant__ float c_tc3[MAX_SLICES];
static float h_tc3[MAX_SLICES];


const int MaxCompositeSlices = MAX_SLICES;

//#define SHMCLASS
//#define NOOP
//#define NOLOAD
//#define NODISPLAY
//#define SHMLOAD
#define VOLTEX3D 3 // undef, 1 or 3
//#define PITCHED
//#define FLOATDATA
//#define CONSTLOAD
//#define THREADPERVOXEL
//#define CONSTDATA

#ifdef VOLTEX3D
#undef PITCHED
#undef CONSTLOAD
#undef CONSTDATA
#undef NOLOAD
#undef SHMLOAD
#undef SHMCLASS
#endif

#ifdef CONSTDATA
#undef SHMLOAD
#endif

#ifdef CONSTLOAD
#undef NOLOAD
#endif

#ifdef NOLOAD
#define SHMLOAD
#endif

#ifdef SHMCLASS
#define SHMLOAD
texture<uchar4, 1, hipReadModeElementType> tex_tf;
#else
texture<uchar4, 1, hipReadModeNormalizedFloat> tex_tf;
#endif

#ifdef VOLTEX3D
texture<uchar4, 2, hipReadModeNormalizedFloat> tex_preint;
#ifdef FLOATDATA
texture<float, 3, hipReadModeElementType> tex_raw;
#else
texture<uchar, 3, hipReadModeNormalizedFloat> tex_raw;
#endif
#endif

#ifdef FLOATDATA
typedef float Scalar;
#else
typedef uchar Scalar;
#endif

typedef void (*CompositionFunction)(
      uchar4 * __restrict__ img, int width, int height,
#ifdef PITCHED
      const hipPitchedPtr pvoxels,
#else
      const Scalar * __restrict__ voxels,
#endif
      int firstSlice, int lastSlice,
      int from, int to);

//----------------------------------------------------------------------------
// device code (CUDA)
//----------------------------------------------------------------------------

__global__ void clearImage(uchar4 * __restrict__ img, int width, int height,
      int from, int to)
{
    const int line = blockIdx.x+from;
    if (line >= to)
        return;

    for (int ix=threadIdx.x; ix<width; ix+=blockDim.x)
    {
        uchar4 *dest = img + line*width+ix;
        *dest = make_uchar4(0, 0, 0, 0);
    }
}

__device__ void blend(uchar4 *dst, float4 src)
{
    uchar4 c = *dst;
    *dst = make_uchar4(src.x * 255.f + c.x*src.w,
            src.y * 255.f + c.y*src.w,
            src.z * 255.f + c.z*src.w,
            (1.f-src.w) * (255-c.w) + c.w);
}

__device__ void blend(uchar4 *dst, uchar4 src)
{
    uchar4 c = *dst;
    const float w = src.w/255.f;
    //src.w = 255 - src.w;
    *dst = make_uchar4(src.x + c.x*w,
            src.y + c.y*w,
            src.z + c.z*w,
            (1.f-w) * (255-c.w) + c.w);
}

__device__ void initPixel(float4 *pix)
{
    *pix = make_float4(0, 0, 0, 1.f);
}

__device__ void initPixel(uchar4 *pix)
{
    *pix = make_uchar4(0, 0, 0, 255);
}

__device__ bool isOpaque(float4 pix)
{
    return (pix.w < 0.003f);
}

__device__ bool isOpaque(uchar4 pix)
{
    return (pix.w < 1);
}

template<typename Pixel>
__device__ Pixel classify(float s)
{
    return tex1Dfetch(tex_tf, s*255.f);
}

template<typename Pixel>
__device__ Pixel classify(uchar s)
{
    return tex1Dfetch(tex_tf, s);
}

template<typename Scalar, int BPV, typename Pixel, int sliceStep, int principal, bool earlyRayTerm>
__global__ void compositeSlicesNearest(
      uchar4 * __restrict__ img, int width, int height,
#ifdef PITCHED
      const hipPitchedPtr pvoxels,
#else
      const Scalar * __restrict__ voxels,
#endif
      int firstSlice, int lastSlice,
      int from, int to)
{
#ifndef VOLTEX3D
#ifdef PITCHED
    const Scalar *voxels = (Scalar *)pvoxels.ptr;
    const int pitch = pvoxels.pitch;
#else
    const int pitch = c_vox[principal] * BPV * sizeof(Scalar);
#endif
#endif
    const int line = blockIdx.x+from;
    if (line >= to)
        return;

    // initialise intermediate image line
    extern __shared__ char smem[];
    Pixel *imgLine = (Pixel *)smem;
#ifdef SHMLOAD
#ifdef SHMCLASS
    uchar4 *voxel = (uchar4 *)(smem+width*sizeof(Pixel));
#else
    Scalar *voxel = (Scalar *)(smem+width*sizeof(Pixel));
#endif
#endif

    for (int ix=threadIdx.x; ix<width; ix+=blockDim.x)
    {
        initPixel(&imgLine[ix]);
    }

    // composite slices for this image line
    for (int slice=firstSlice; slice!=lastSlice; slice += sliceStep)
    {
#ifdef CONSTLOAD
        const Scalar *voxLine = (Scalar *)(((uchar *)voxels) + pitch * c_vox[principal+1]);
#endif
#ifdef NOOP
        const int iPosY = line;
#else
        // compute upper left image corner
        const int iPosY = c_start[slice].y;

        if(line < iPosY)
            continue;
        if(line >= iPosY+c_vox[principal+1])
            continue;

        const int iPosX = c_start[slice].x;
#endif

        // the voxel row of the current slice corresponding to this image line
#ifndef NOLOAD
#ifndef CONSTLOAD
#ifndef VOLTEX3D
        const Scalar *voxLine = (Scalar *)(((uchar *)voxels) + pitch * ((slice+1)*c_vox[principal+1] + (iPosY-line-1)));
#endif
#endif

#ifdef SHMLOAD
        for (int ix=threadIdx.x; ix<c_vox[principal+0]; ix+=blockDim.x)
        {
#ifdef SHMCLASS
            voxel[ix] = classify<uchar4>(voxLine[ix]);
#else
            voxel[ix] = voxLine[ix];
#endif
        }
        __syncthreads();
#endif
#endif

#ifndef NOOP
        // Traverse intermediate image pixels which correspond to the current slice.
        // 1 is subtracted from each loop counter to remain inside of the volume boundaries:
#ifdef THREADPERVOXEL
        for (int ix=threadIdx.x; ix<c_vox[principal+0]; ix+=blockDim.x)
#else
        for (int ix=threadIdx.x; ix<c_vox[principal+0]+iPosX; ix+=blockDim.x)
#endif
        {
#ifndef THREADPERVOXEL
            if(ix<iPosX)
                continue;
#endif
#ifdef THREADPERVOXEL
            const int vidx = ix;
            const int iidx = ix + iPosX;
#else
            const int vidx = ix - iPosX;
            const int iidx = ix;
#endif

            // pointer to destination pixel
            Pixel *pix = imgLine + iidx;
            Pixel d = *pix;
            if(earlyRayTerm && isOpaque(d))
                continue;

#ifdef VOLTEX3D
#if VOLTEX3D == 3
            const float v = tex3D(tex_raw, vidx, c_vox[principal+1]+iPosY-line-1, slice);
#else
            float v;
            switch(principal)
            {
                case 0:
                    v = tex3D(tex_raw, c_vox[2]-slice-1, c_vox[0]-vidx-1, c_vox[1]+iPosY-line-1);
                    break;
                case 1:
                    v = tex3D(tex_raw, line-iPosY, slice, c_vox[1]-vidx-1);
                    break;
                case 2:
                    v = tex3D(tex_raw, vidx, c_vox[0]+iPosY-line-1, slice);
                    break;
            }
#endif
            const float4 c = classify<float4>(v);
#else
#ifdef CONSTDATA
            const float4 c = classify<float4>(uchar(ix));
#else
#ifdef SHMCLASS
            const uchar4 v = *(voxel + BPV * vidx);
            const float4 c = make_float4(v.x/255.f, v.y/255.f, v.z/255.f, v.w/255.f);
#else
            // fetch scalar voxel value
#ifdef SHMLOAD
            const Scalar *v = voxel + BPV * vidx;
#else
            const Scalar *v = voxLine + BPV * vidx;
#endif
            // apply transfer function
            const float4 c = classify<float4>(*v);
#endif
#endif
#endif

            // blend
            const float w = d.w*c.w;
            d.x += w*c.x;
            d.y += w*c.y;
            d.z += w*c.z;
            d.w -= w;

            // store into shmem
            *pix = d;
#ifdef THREADPERVOXEL
            __syncthreads();
#endif
        }
#endif
    }

#ifndef NODISPLAY
    // copy line to intermediate image
    for (int ix=threadIdx.x; ix<width; ix+=blockDim.x)
    {
        uchar4 *dest = img + line*width+ix;
        blend(dest, imgLine[ix]);
    }
#endif
}


#ifdef VOLTEX3D
template<typename Scalar, int BPV, typename Pixel, int sliceStep, int principal, bool earlyRayTerm>
__global__ void compositeSlicesBilinear(
      uchar4 * __restrict__ img, int width, int height,
#ifdef PITCHED
      const hipPitchedPtr pvoxels,
#else
      const Scalar * __restrict__ voxels,
#endif
      int firstSlice, int lastSlice,
      int from, int to)
{
    const int line = blockIdx.x+from;
    if (line >= to)
        return;

    // initialise intermediate image line
    extern __shared__ char smem[];
    Pixel *imgLine = (Pixel *)smem;

    for (int ix=threadIdx.x; ix<width; ix+=blockDim.x)
    {
        initPixel(&imgLine[ix]);
    }

    // composite slices for this image line
    for (int slice=firstSlice; slice!=lastSlice; slice += sliceStep)
    {
        // compute upper left image corner
        const int iPosY = c_start[slice].y;

        if(line < iPosY)
            continue;
        if(line >= iPosY+c_vox[principal+1])
            continue;

        const int iPosX = c_start[slice].x;

        // Traverse intermediate image pixels which correspond to the current slice.
        // 1 is subtracted from each loop counter to remain inside of the volume boundaries:
        for (int ix=threadIdx.x; ix<c_vox[principal+0]+iPosX; ix+=blockDim.x)
        {
            if(ix<iPosX)
                continue;
            const int vidx = ix - iPosX;
            const int iidx = ix;

            // pointer to destination pixel
            Pixel *pix = imgLine + iidx;
            Pixel d = *pix;
            if(earlyRayTerm && isOpaque(d))
                continue;

            const float x = c_tcStart[slice].x + c_tcStep[slice].x*vidx;
            const float y = c_tcStart[slice].y + c_tcStep[slice].y*(line-iPosY);
            const float z = c_tc3[slice];
#if VOLTEX3D==3
            const float v = tex3D(tex_raw, x, y, z);
#else
            float v;
            switch(principal)
            {
                case 0:
                    v = tex3D(tex_raw, z, x, y);
                    break;
                case 1:
                    v = tex3D(tex_raw, y, z, x);
                    break;
                case 2:
                    v = tex3D(tex_raw, x, y, z);
                    break;
            }
#endif
            const float4 c = classify<float4>(v);

            // blend
            const float w = d.w*c.w;
            d.x += w*c.x;
            d.y += w*c.y;
            d.z += w*c.z;
            d.w -= w;

            // store into shmem
            *pix = d;
        }
    }

    // copy line to intermediate image
    for (int ix=threadIdx.x; ix<width; ix+=blockDim.x)
    {
        uchar4 *dest = img + line*width+ix;
        blend(dest, imgLine[ix]);
    }
}

template<typename Scalar, int BPV, typename Pixel, int sliceStep, int principal, bool earlyRayTerm>
__global__ void compositeSlicesPreIntegrated(
      uchar4 * __restrict__ img, int width, int height,
#ifdef PITCHED
      const hipPitchedPtr pvoxels,
#else
      const Scalar * __restrict__ voxels,
#endif
      int firstSlice, int lastSlice,
      int from, int to)
{
    const int line = blockIdx.x+from;
    if (line >= to)
        return;

    // initialise intermediate image line
    extern __shared__ char smem[];
    Pixel *imgLine = (Pixel *)smem;
    Scalar *sf = (Scalar *)&imgLine[width];

    for (int ix=threadIdx.x; ix<width; ix+=blockDim.x)
    {
        initPixel(&imgLine[ix]);
        sf[ix] = 0.f;
    }

    // composite slices for this image line
    for (int slice=firstSlice; slice!=lastSlice; slice += sliceStep)
    {
        // compute upper left image corner
        const int iPosY = c_start[slice].y;

        if(line < iPosY)
            continue;
        if(line >= iPosY+c_vox[principal+1])
            continue;

        const int iPosX = c_start[slice].x;

        // Traverse intermediate image pixels which correspond to the current slice.
        // 1 is subtracted from each loop counter to remain inside of the volume boundaries:
        for (int ix=threadIdx.x; ix<c_vox[principal+0]+iPosX; ix+=blockDim.x)
        {
            if(ix<iPosX)
                continue;
            const int vidx = ix - iPosX;
            const int iidx = ix;

            // pointer to destination pixel
            Pixel *pix = imgLine + iidx;
            Pixel d = *pix;
            if(earlyRayTerm && isOpaque(d))
                continue;

            const float x = c_tcStart[slice].x + c_tcStep[slice].x*vidx;
            const float y = c_tcStart[slice].y + c_tcStep[slice].y*(line-iPosY);
            const float z = c_tc3[slice];
#if VOLTEX3D==3
            const float sb = tex3D(tex_raw, x, y, z);
#else
            float sb;
            switch(principal)
            {
                case 0:
                    sb = tex3D(tex_raw, z, x, y);
                    break;
                case 1:
                    sb = tex3D(tex_raw, y, z, x);
                    break;
                case 2:
                    sb = tex3D(tex_raw, x, y, z);
                    break;
            }
#endif
            if(slice != firstSlice)
            {
                const float4 c = tex2D(tex_preint, sf[ix], sb);

                // blend
                const float w = d.w*c.w;
                d.x += w*c.x;
                d.y += w*c.y;
                d.z += w*c.z;
                d.w -= w;

                // store into shmem
                *pix = d;
            }
            sf[ix] = sb;
        }
    }

    // copy line to intermediate image
    for (int ix=threadIdx.x; ix<width; ix+=blockDim.x)
    {
        uchar4 *dest = img + line*width+ix;
        blend(dest, imgLine[ix]);
    }
}
#endif

//----------------------------------------------------------------------------
// host code
//----------------------------------------------------------------------------


//----------------------------------------------------------------------------
/** Constructor.
  @param vd volume description of volume to display
  @see vvRenderer
*/
vvCudaPar::vvCudaPar(vvVolDesc* vd, vvRenderState rs) : vvSoftPar(vd, rs)
{
   vvDebugMsg::msg(1, "vvCudaPar::vvCudaPar()");

   rendererType = CUDAPAR;
   imagePrecision = 8;
   earlyRayTerm = true;
   mappedImage = false;
   if (warpMode==TEXTURE)
   {
       // we need a power-of-2 image size for glTexImage2D
       int imgSize = vvToolshed::getTextureSize(2 * ts_max(vd->vox[0], vd->vox[1], vd->vox[2]));

#ifndef NODISPLAY
       if (vvCuda::initGlInterop())
       {
           vvDebugMsg::msg(1, "using CUDA/GL interop");
           // avoid image copy from GPU to CPU and back
           setWarpMode(CUDATEXTURE);
           intImg->setSize(imgSize, imgSize, NULL, true);
       }
       else
       {
         vvDebugMsg::msg(1, "can't use CUDA/GL interop");
         intImg->setSize(imgSize, imgSize);
       }
#endif
   }

   wViewDir.set(0.0f, 0.0f, 1.0f);

#ifdef FLOATDATA
   for(int i=0; i<3; ++i)
   {
       size_t vox = vd->vox[0]*vd->vox[1]*vd->vox[2];
       fraw[i] = new float[vox];
       for(size_t j=0; j<vox; ++j)
       {
           fraw[i][j] = raw[i][j] / 255.f;
       }
   }
#endif

   bool ok = true;
#if defined(PITCHED) || defined(VOLTEX3D)
#ifdef VOLTEX3D
   for (int i=0; i<VOLTEX3D; ++i)
#else
   for (int i=0; i<3; ++i)
#endif
   {
#ifdef PITCHED
       hipExtent extent = make_hipExtent(vd->vox[(i+1)%3]*sizeof(Scalar), vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
       if(!vvCuda::checkError(&ok, hipMalloc3D(&d_voxptr[i], extent), "hipMalloc3D vox"))
           break;
#else
#if defined(VOLTEX3D) && VOLTEX3D == 1
       hipExtent extent = make_hipExtent(vd->vox[0], vd->vox[1], vd->vox[2]);
#else
       hipExtent extent = make_hipExtent(vd->vox[(i+1)%3], vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
#endif
       hipChannelFormatDesc desc = hipCreateChannelDesc<Scalar>();
       if(!vvCuda::checkError(&ok, hipMalloc3DArray(&d_voxarr[i], &desc, extent, 0), "hipMalloc3DArray vox"))
           break;
#endif
       hipMemcpy3DParms parms = {0};
#if defined(VOLTEX3D) && VOLTEX3D == 1
#ifdef FLOATDATA
       parms.srcPtr = make_hipPitchedPtr(fraw[2], sizeof(Scalar)*vd->vox[0], vd->vox[0], vd->vox[1]);
#else
       parms.srcPtr = make_hipPitchedPtr(raw[2], sizeof(Scalar)*vd->vox[0], vd->vox[0], vd->vox[1]);
#endif
#else
#ifdef FLOATDATA
       parms.srcPtr = make_hipPitchedPtr(fraw[i], sizeof(Scalar)*vd->vox[(i+1)%3], vd->vox[(i+1)%3], vd->vox[(i+2)%3]);
#else
       parms.srcPtr = make_hipPitchedPtr(raw[i], sizeof(Scalar)*vd->vox[(i+1)%3], vd->vox[(i+1)%3], vd->vox[(i+2)%3]);
#endif
#endif

#ifdef PITCHED
       parms.dstPtr = d_voxptr[i];
       parms.extent = make_hipExtent(vd->vox[(i+1)%3]*sizeof(Scalar), vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
#else
       parms.dstArray = d_voxarr[i];
#if defined(VOLTEX3D) && VOLTEX3D == 1
       parms.extent = make_hipExtent(vd->vox[0], vd->vox[1], vd->vox[2]);
#else
       parms.extent = make_hipExtent(vd->vox[(i+1)%3], vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
#endif
#endif
       parms.kind = hipMemcpyHostToDevice;
       if(!vvCuda::checkError(&ok, hipMemcpy3D(&parms), "hipMemcpy3D vox"))
           break;
   }
#else
   // alloc memory for voxel arrays (for each principal viewing direction)
   vvCuda::checkError(&ok, hipMalloc(&d_voxels, sizeof(Scalar)*vd->vox[0]*vd->vox[1]*vd->vox[2]*3), "hipMalloc vox");
   for (int i=0; i<3; ++i)
   {
#ifdef FLOATDATA
       if (!vvCuda::checkError(&ok, hipMemcpy(d_voxels+i*sizeof(Scalar)*vd->vox[0]*vd->vox[1]*vd->vox[2],
                   fraw[i], sizeof(Scalar)*vd->getFrameBytes(), hipMemcpyHostToDevice), "hipMemcpy vox"))
#else
       if (!vvCuda::checkError(&ok, hipMemcpy(d_voxels+i*sizeof(Scalar)*vd->vox[0]*vd->vox[1]*vd->vox[2],
                   raw[i], vd->getFrameBytes(), hipMemcpyHostToDevice), "hipMemcpy vox"))
#endif
          break;
   }
#endif

   // transfer function is stored as a texture
   vvCuda::checkError(&ok, hipMalloc(&d_tf, 4096*4), "hipMalloc tf");
   vvCuda::checkError(&ok, hipBindTexture(NULL, tex_tf, d_tf, 4096), "bind tf tex");

   // pre-integration table
   hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
   vvCuda::checkError(&ok, hipMallocArray(&d_preint, &desc, PRE_INT_TABLE_SIZE, PRE_INT_TABLE_SIZE), "hipMalloc preint");
   tex_preint.normalized = true;
   tex_preint.filterMode = bilinLookup ? hipFilterModeLinear : hipFilterModePoint;
   tex_preint.addressMode[0] = hipAddressModeClamp;
   tex_preint.addressMode[1] = hipAddressModeClamp;
   vvCuda::checkError(&ok, hipBindTextureToArray(tex_preint, d_preint, desc), "bind preint tex");

#ifndef NODISPLAY
   // allocate output image (intermediate image)
   if (warpMode==CUDATEXTURE)
   {
       vvCuda::checkError(&ok, hipGraphicsGLRegisterBuffer(&intImgRes, intImg->getPboName(), cudaGraphicsMapFlagsWriteDiscard), "map PBO to CUDA");
   }
   else if (mappedImage)
   {
       vvCuda::checkError(&ok, hipHostAlloc(&h_img, intImg->width*intImg->height*vvSoftImg::PIXEL_SIZE, hipHostMallocMapped), "img alloc");;
       intImg->setSize(intImg->width, intImg->height, h_img, false);
       vvCuda::checkError(&ok, hipHostGetDevicePointer(&d_img, h_img, 0), "get dev ptr img");
   }
   else
#endif
   {
       vvCuda::checkError(&ok, hipMalloc(&d_img, intImg->width*intImg->height*vvSoftImg::PIXEL_SIZE), "hipMalloc img");
   }

   // copy volume size (in voxels)
   int h_vox[5];
   for (int i=0; i<5; ++i)
       h_vox[i] = vd->vox[(i+1)%3];
   vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_vox), h_vox, sizeof(int)*5), "hipMemcpy vox");

   updateTransferFunction();
}


//----------------------------------------------------------------------------
/// Destructor.
vvCudaPar::~vvCudaPar()
{
   vvDebugMsg::msg(1, "vvCudaPar::~vvCudaPar()");

#ifdef FLOATDATA
   for(int i=0; i<3; ++i)
     delete[] fraw[i];
#endif

#ifndef NODISPLAY
   if (warpMode==CUDATEXTURE)
      hipGraphicsUnregisterResource(intImgRes);
   else if (mappedImage)
       hipHostFree(h_img);
   else
#endif
       hipFree(d_img);

   hipUnbindTexture(tex_tf);
   hipFree(d_tf);

   hipUnbindTexture(tex_preint);
   hipFree(d_preint);
#ifdef VOLTEX3D
   for(int i=0; i<VOLTEX3D; ++i)
     hipFree(d_voxarr[i]);
#else
#ifdef PITCHED
   for(int i=0; i<3; ++i)
       hipFree(d_voxptr[i].ptr);
#else
   hipFree(d_voxels);
#endif
#endif
}

void vvCudaPar::updateTransferFunction()
{
   vvDebugMsg::msg(2, "vvCudaPar::updateTransferFunction()");

   vvSoftPar::updateTransferFunction();

   vvCuda::checkError(NULL, hipMemcpy(d_tf, rgbaConv, sizeof(rgbaConv), hipMemcpyHostToDevice), "hipMemcpy tf");
   if(preIntegration)
   {
       vvCuda::checkError(NULL, hipMemcpyToArray(d_preint, 0, 0, &preIntTable[0][0][0],
                   PRE_INT_TABLE_SIZE*PRE_INT_TABLE_SIZE*4, hipMemcpyHostToDevice), "hipMemcpy preint");
   }
}

template<typename Pixel, int principal, int sliceStep, bool earlyRayTerm>
CompositionFunction selectComposition(vvCudaPar *rend)
{
#ifdef VOLTEX3D
    if(rend->getSliceInterpol())
    {
        if(rend->getPreIntegration())
            return compositeSlicesPreIntegrated<Scalar, 1, Pixel, sliceStep, principal, earlyRayTerm>;
        else
            return compositeSlicesBilinear<Scalar, 1, Pixel, sliceStep, principal, earlyRayTerm>;
    }
    else
#endif
        return compositeSlicesNearest<Scalar, 1, Pixel, sliceStep, principal, earlyRayTerm>;
}

template<typename Pixel, int principal, int sliceStep>
CompositionFunction selectCompositionWithEarlyTermination(vvCudaPar *rend)
{
    if(rend->getEarlyRayTerm())
        return selectComposition<Pixel, principal, sliceStep, true>(rend);
    else
        return selectComposition<Pixel, principal, sliceStep, false>(rend);
}

template<typename Pixel, int principal>
CompositionFunction selectCompositionWithSliceStep(vvCudaPar *rend, int sliceStep)
{
    switch(sliceStep)
    {
        case 1:
            return selectCompositionWithEarlyTermination<Pixel, principal,1>(rend);
        case -1:
            return selectCompositionWithEarlyTermination<Pixel, principal,-1>(rend);
        default:
            assert("slice step out of range" == NULL);
    }

    return NULL;
}

template<typename Pixel>
CompositionFunction selectCompositionWithPrincipal(vvCudaPar *rend, int sliceStep)
{
    switch(rend->getPrincipal())
    {
        case 0:
            return selectCompositionWithSliceStep<Pixel, 0>(rend, sliceStep);
        case 1:
            return selectCompositionWithSliceStep<Pixel, 1>(rend, sliceStep);
        case 2:
            return selectCompositionWithSliceStep<Pixel, 2>(rend, sliceStep);
        default:
            assert("principal axis out of range" == NULL);

    }

    return NULL;
}

CompositionFunction selectCompositionWithPrecision(vvCudaPar *rend, int sliceStep)
{
    switch(rend->getPrecision())
    {
        case 8:
            return selectCompositionWithPrincipal<uchar4>(rend, sliceStep);
        case 32:
            return selectCompositionWithPrincipal<float4>(rend, sliceStep);
        default:
            assert("invalid precision" == NULL);
    }

    return NULL;
}

//----------------------------------------------------------------------------
/** Composite the volume slices to the intermediate image.
  The function prepareRendering() must be called before this method.
  The shear transformation matrices have to be computed before calling this method.
  The volume slices are processed from front to back.
  @param from,to optional arguments to define first and last intermediate image line to render.
                 if not passed, the entire intermediate image will be rendered
*/
void vvCudaPar::compositeVolume(int from, int to)
{
   vvDebugMsg::msg(3, "vvCudaPar::compositeVolume(): ", from, to);

   // If stacking==true then draw front to back, else draw back to front:
   int firstSlice = (stacking) ? 0 : (len[2]-1);  // first slice to process
   int lastSlice  = (stacking) ? (len[2]-1) : 0;  // last slice to process
   int sliceStep  = (stacking) ? 1 : -1;          // step size to get to next slice

   earlyRayTermination = 0;

   if (from == -1)
       from = 0;
   if (to == -1)
       to = intImg->height;

   // compute data for determining upper left image corner of each slice and copy it to device
   vvVector4 start, end;
   findSlicePosition(firstSlice, &start, &end);
   vvVector4 sinc, einc;
   findSlicePosition(firstSlice+sliceStep, &sinc, &einc);
   sinc.sub(&start);
   einc.sub(&end);
   vvVector4 scur = start;
   vvVector4 ecur = end;
#if defined(VOLTEX3D) && VOLTEX3D==1
   const int p = principal;
#else
   const int p = 2;
#endif
   for(int slice=firstSlice; slice != lastSlice; slice += sliceStep)
   {
#ifdef VOLTEX3D
       if(sliceInterpol)
       {
           const float sx = scur.e[0]/scur.e[3];
           const float sy = scur.e[1]/scur.e[3];
           const float ex = ecur.e[0]/ecur.e[3];
           const float ey = ecur.e[1]/ecur.e[3];

           h_start[slice].x = int(floor(sx));
           h_start[slice].y = int(floor(sy));

           h_stop[slice].x = int(ceil(ex));
           h_stop[slice].y = int(ceil(ey));

           switch(p)
           {
               case 0:
                   h_tcStep[slice].x = -1.f/(ex-sx);
                   h_tcStep[slice].y = -1.f/(ey-sy);

                   h_tcStart[slice].x = 1.f + (h_start[slice].x - sx + 0.5f)*h_tcStep[slice].x;
                   h_tcStart[slice].y = 1.f + (h_start[slice].y - sy + 0.5f)*h_tcStep[slice].y;

                   h_tc3[slice] = 1.f-(slice+0.5f)*1.f/vd->vox[principal];
                   break;
                case 1:
                   h_tcStep[slice].x = -1.f/(ex-sx);
                   h_tcStep[slice].y = 1.f/(ey-sy);

                   h_tcStart[slice].x = 1.f + (h_start[slice].x - sx + 0.5f)*h_tcStep[slice].x;
                   h_tcStart[slice].y = (h_start[slice].y - sy + 0.5f)*h_tcStep[slice].y;

                   h_tc3[slice] = (slice+0.5f)*1.f/vd->vox[principal];
                   break;
                case 2:
                   h_tcStep[slice].x = 1.f/(ex-sx);
                   h_tcStep[slice].y = -1.f/(ey-sy);

                   h_tcStart[slice].x = (h_start[slice].x - sx + 0.5f)*h_tcStep[slice].x;
                   h_tcStart[slice].y = 1.f + (h_start[slice].y - sy + 0.5f)*h_tcStep[slice].y;

                   h_tc3[slice] = (slice+0.5f)*1.f/vd->vox[principal];
                   break;
           }

           ecur.add(&sinc);
       }
       else
#endif
       {
           h_start[slice].x = int(scur.e[0] / scur.e[3] + 0.5f);
           h_start[slice].y = int(scur.e[1] / scur.e[3] + 0.5f);
       }
       scur.add(&sinc);
   }

   bool ok = true;
   vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_start), h_start, sizeof(h_start)), "hipMemcpy start");
#ifdef VOLTEX3D
   if(sliceInterpol)
   {
       vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_stop), h_stop, sizeof(h_stop)), "hipMemcpy stop");
       vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_tcStart), h_tcStart, sizeof(h_tcStart)), "hipMemcpy tcStart");
       vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_tcStep), h_tcStep, sizeof(h_tcStep)), "hipMemcpy tcStep");
       vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_tc3), h_tc3, sizeof(h_tc3)), "hipMemcpy tc3");
   }
#endif

#ifdef VOLTEX3D
   tex_raw.normalized = sliceInterpol;
   tex_raw.filterMode = sliceInterpol ? hipFilterModeLinear : hipFilterModePoint;
   tex_raw.addressMode[0] = hipAddressModeClamp;
   tex_raw.addressMode[1] = hipAddressModeClamp;
   tex_raw.addressMode[2] = hipAddressModeClamp;
   hipChannelFormatDesc desc = hipCreateChannelDesc<Scalar>();
#if VOLTEX3D == 1
   hipBindTextureToArray(tex_raw, d_voxarr[0], desc);
#else
   hipBindTextureToArray(tex_raw, d_voxarr[principal], desc);
#endif
#endif

#ifndef NODISPLAY
   // prepare intermediate image
   if (warpMode==CUDATEXTURE)
   {
       vvCuda::checkError(&ok, hipGraphicsMapResources(1, &intImgRes, NULL), "map CUDA resource");
       size_t size;
       vvCuda::checkError(&ok, hipGraphicsResourceGetMappedPointer((void**)&d_img, &size, intImgRes), "get PBO mapping");
       assert(size == intImg->width*intImg->height*vvSoftImg::PIXEL_SIZE);
   }
   else
   {
       intImg->clear();
   }
#endif

   int shmsize = intImg->width*imagePrecision/8*4;
#ifdef SHMLOAD
   shmsize += vd->vox[principal]*vd->getBPV()*sizeof(Scalar);
#endif
   if(preIntegration)
   {
       shmsize += intImg->width*sizeof(Scalar);
   }

   clearImage <<<to-from, 128, shmsize>>>(d_img, intImg->width, intImg->height, from, to);

   CompositionFunction compose = selectCompositionWithPrecision(this, sliceStep);
   // do the computation on the device
   for(int i=lastSlice; i*sliceStep>firstSlice*sliceStep; i-=sliceStep*MaxCompositeSlices)
   {
       hipDeviceSynchronize();
#ifdef PITCHED
       compose <<<to-from, 128, shmsize>>>(
               d_img, intImg->width, intImg->height,
               d_voxptr[principal],
               sliceStep*max(sliceStep*i-MaxCompositeSlices, sliceStep*firstSlice), i,
               from, to);
#else
       compose <<<to-from, 128, shmsize>>>(
               d_img, intImg->width, intImg->height,
               (Scalar *)(d_voxels+sizeof(Scalar)*vd->getBPV()*principal*(vd->vox[0]*vd->vox[1]*vd->vox[2])),
               sliceStep*max(sliceStep*i-MaxCompositeSlices, sliceStep*firstSlice), i,
               from, to);
#endif
   }

#ifdef VOLTEX3D
   hipUnbindTexture(tex_raw);
#endif

   // copy back or unmap for using as PBO
   ok = vvCuda::checkError(&ok, hipGetLastError(), "start kernel");
#ifndef NODISPLAY
   if (warpMode==CUDATEXTURE)
   {
       vvCuda::checkError(&ok, hipGraphicsUnmapResources(1, &intImgRes, NULL), "unmap CUDA resource");
   }
   else if (mappedImage)
   {
       hipDeviceSynchronize();
   }
   else
   {
       hipMemcpy(intImg->data, d_img, intImg->width*intImg->height*vvSoftImg::PIXEL_SIZE, hipMemcpyDeviceToHost);
       ok = vvCuda::checkError(&ok, hipGetLastError(), "cpy to host");
   }
#endif
}

void vvCudaPar::setParameter(ParameterType param, float val, char *cval)
{
    vvDebugMsg::msg(3, "vvCudaPar::setParameter()");
    switch(param)
    {
        case VV_IMG_PRECISION:
            if(val == 8)
                imagePrecision = 8;
            else
                imagePrecision = 32;
            break;
        case VV_TERMINATEEARLY:
            earlyRayTerm = (val != 0.f);
            break;
        default:
            vvSoftPar::setParameter(param, val, cval);
            break;
    }
}

float vvCudaPar::getParameter(ParameterType param, char *cval) const
{
    vvDebugMsg::msg(3, "vvCudaPar::getParameter()");
    switch(param)
    {
        case VV_IMG_PRECISION:
            return imagePrecision;
        case VV_TERMINATEEARLY:
            return (earlyRayTerm ? 1.f : 0.f);
        default:
            return vvSoftPar::getParameter(param, cval);
    }
}

//============================================================================
// End of File
//============================================================================
