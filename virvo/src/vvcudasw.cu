#include "hip/hip_runtime.h"
// Virvo - Virtual Reality Volume Rendering
// Copyright (C) 2010 University of Cologne
// Contact: Jurgen P. Schulze, jschulze@ucsd.edu
//
// This file is part of Virvo.
//
// Virvo is free software; you can redistribute it and/or
// modify it under the terms of the GNU Lesser General Public
// License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
//
// This library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public
// License along with this library (see license.txt); if not, write to the 
// Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA

#include <iostream>
using std::cerr;
using std::endl;

#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include "vvdebugmsg.h"
#include "vvvecmath.h"
#include "vvstopwatch.h"
#include "vvcuda.h"
#include "vvcudaimg.h"
#include "vvcudasw.h"

const int MAX_SLICES = 1600;
const int SliceStack = 32;

__constant__ int   c_vox[5];
__constant__ int2 c_start[MAX_SLICES];
static int2 h_start[MAX_SLICES];
__constant__ int2 c_stop[MAX_SLICES];
static int2 h_stop[MAX_SLICES];
__constant__ float2 c_tcStart[MAX_SLICES];
static float2 h_tcStart[MAX_SLICES];
__constant__ float2 c_tcStep[MAX_SLICES];
static float2 h_tcStep[MAX_SLICES];
__constant__ float c_tc3[MAX_SLICES];
static float h_tc3[MAX_SLICES];
__constant__ float c_zStep;


const int MaxCompositeSlices = MAX_SLICES;

const int nthreads = 128;
const dim3 Patch(16, 16);

#define VARSTEP
#define PATCHES
//#define UNROLL
//#define NOSHMEM
//#define SHMCLASS
//#define NOOP
//#define NOLOAD
//#define NODISPLAY
//#define SHMLOAD
#define VOLTEX3D 3 // undef, 1 or 3
//#define PITCHED
//#define FLOATDATA
//#define CONSTLOAD
//#define THREADPERVOXEL
//#define CONSTDATA

#ifdef PATCHES
#define NOSHMEM
#endif

#ifdef VOLTEX3D
#undef PITCHED
#undef CONSTLOAD
#undef CONSTDATA
#undef NOLOAD
#undef SHMLOAD
#undef SHMCLASS
#endif

#ifdef CONSTDATA
#undef SHMLOAD
#endif

#ifdef CONSTLOAD
#undef NOLOAD
#endif

#ifdef NOLOAD
#define SHMLOAD
#endif

typedef uchar4 LutEntry;

#ifdef SHMCLASS
#define SHMLOAD
texture<LutEntry, 1, hipReadModeElementType> tex_tf;
#else
texture<LutEntry, 1, hipReadModeNormalizedFloat> tex_tf;
#endif

texture<LutEntry, 2, hipReadModeNormalizedFloat> tex_preint;
texture<uchar, 2, hipReadModeNormalizedFloat> tex_minmaxTable;

#ifdef VOLTEX3D
#ifdef FLOATDATA
texture<float, 3, hipReadModeElementType> tex_raw;
#else
texture<uchar, 3, hipReadModeNormalizedFloat> tex_raw;
#endif
texture<uchar, 3, hipReadModeNormalizedFloat> tex_min;
texture<uchar, 3, hipReadModeNormalizedFloat> tex_max;
#endif

#ifdef FLOATDATA
typedef float Scalar;
#else
typedef uchar Scalar;
#endif

typedef void (*CompositionFunction)(
      uchar4 * __restrict__ img, int width, int height,
#ifdef PITCHED
      const hipPitchedPtr pvoxels,
#else
      const Scalar * __restrict__ voxels,
#endif
      int firstSlice, int lastSlice,
      int2 from, int2 to, int nslice, float scale);

//----------------------------------------------------------------------------
// device code (CUDA)
//----------------------------------------------------------------------------

__device__ int2 coord(int2 from)
{
    return make_int2(threadIdx.x + blockDim.x*blockIdx.x + from.x,
            threadIdx.y + blockDim.y*blockIdx.y + from.y);
}

__global__ void clearImage(uchar4 * __restrict__ img, int width, int height,
      int fromY, int toY)
{
    const int2 p = coord(make_int2(0, fromY));
    if (p.y >= toY)
        return;
    if (p.x >= width)
        return;

    uchar4 *dest = img + p.y*width+p.x;
    *dest = make_uchar4(0, 0, 0, 0);
}

__device__ void blend(uchar4 *dst, float4 src)
{
    uchar4 c = *dst;
    *dst = make_uchar4(src.x * 255.f + c.x*src.w,
            src.y * 255.f + c.y*src.w,
            src.z * 255.f + c.z*src.w,
            (1.f-src.w) * (255-c.w) + c.w);
}

__device__ void blend(uchar4 *dst, uchar4 src)
{
    uchar4 c = *dst;
    const float w = src.w/255.f;
    *dst = make_uchar4(src.x + c.x*w,
            src.y + c.y*w,
            src.z + c.z*w,
            (1.f-w) * (255-c.w) + c.w);
}

__device__ void initPixel(float4 *pix)
{
    *pix = make_float4(0, 0, 0, 1.f);
}

__device__ void initPixel(uchar4 *pix)
{
    *pix = make_uchar4(0, 0, 0, 255);
}

__device__ bool isOpaque(float4 pix)
{
    return (pix.w < 0.05f);
}

__device__ bool isOpaque(uchar4 pix)
{
    return (pix.w < 13);
}

template<typename Pixel>
__device__ Pixel classify(float s)
{
    return tex1Dfetch(tex_tf, s*255.f);
}

template<typename Pixel>
__device__ Pixel classify(uchar s)
{
    return tex1Dfetch(tex_tf, s);
}

#ifdef VOLTEX3D
__device__ float2 minmax(float x, float y, float z, int principal)
{
    switch(principal)
    {
        case 0:
            return make_float2(tex3D(tex_min, z, x, y),
                    tex3D(tex_max, z, x, y));
        case 1:
            return make_float2(tex3D(tex_min, y, z, x),
                    tex3D(tex_max, y, z, x));
        case 2:
            return make_float2(tex3D(tex_min, x, y, z),
                    tex3D(tex_max, x, y, z));
    }
    return make_float2(-1.f, -1.f);
}

__device__ float volume(float x, float y, float z, int principal)
{
#if VOLTEX3D==3
    return tex3D(tex_raw, x, y, z);
#else
    switch(principal)
    {
        case 0:
            return tex3D(tex_raw, z, x, y);
        case 1:
            return tex3D(tex_raw, y, z, x);
        case 2:
            return tex3D(tex_raw, x, y, z);
    }
    return -1.f;
#endif
}

__device__ float volume(int px, int py, int slice, int principal)
{
    const float x = c_tcStart[slice].x + c_tcStep[slice].x*px;
    const float y = c_tcStart[slice].y + c_tcStep[slice].y*py;
    const float z = c_tc3[slice];
#if VOLTEX3D==3
    return tex3D(tex_raw, x, y, z);
#else
    switch(principal)
    {
        case 0:
            return tex3D(tex_raw, z, x, y);
        case 1:
            return tex3D(tex_raw, y, z, x);
        case 2:
            return tex3D(tex_raw, x, y, z);
    }
    return -1.f;
#endif
}
#endif

template<typename Scalar, int BPV, typename Pixel, int sliceStep, int principal, bool earlyRayTerm>
__global__ void compositeSlicesNearest(
      uchar4 * __restrict__ img, int width, int height,
#ifdef PITCHED
      const hipPitchedPtr pvoxels,
#else
      const Scalar * __restrict__ voxels,
#endif
      int firstSlice, int lastSlice,
      int2 from, int2 to, int nslice, float sclae)
{
#ifndef VOLTEX3D
#ifdef PITCHED
    const Scalar *voxels = (Scalar *)pvoxels.ptr;
    const int pitch = pvoxels.pitch;
#else
    const int pitch = c_vox[principal] * BPV * sizeof(Scalar);
#endif
#endif
    const int line = blockIdx.x+from.y;
    if (line >= to.y)
        return;

    // initialise intermediate image line
    extern __shared__ char smem[];
    Pixel *imgLine = (Pixel *)smem;
#ifdef SHMLOAD
#ifdef SHMCLASS
    uchar4 *voxel = (uchar4 *)(smem+width*sizeof(Pixel));
#else
    Scalar *voxel = (Scalar *)(smem+width*sizeof(Pixel));
#endif
#endif

    for (int ix=threadIdx.x+from.x; ix<to.x; ix+=blockDim.x)
    {
        initPixel(&imgLine[ix]);
    }

    // composite slices for this image line
    for (int slice=firstSlice; slice!=lastSlice; slice += sliceStep)
    {
#ifdef CONSTLOAD
        const Scalar *voxLine = (Scalar *)(((uchar *)voxels) + pitch * c_vox[principal+1]);
#endif
#ifdef NOOP
        const int iPosY = line;
#else
        // compute upper left image corner
        const int iPosY = c_start[slice].y;

        if(line < iPosY)
            continue;
        if(line >= iPosY+c_vox[principal+1])
            continue;

        const int iPosX = c_start[slice].x;
#endif

        // the voxel row of the current slice corresponding to this image line
#ifndef NOLOAD
#ifndef CONSTLOAD
#ifndef VOLTEX3D
        const Scalar *voxLine = (Scalar *)(((uchar *)voxels) + pitch * ((slice+1)*c_vox[principal+1] + (iPosY-line-1)));
#endif
#endif

#ifdef SHMLOAD
        for (int ix=threadIdx.x; ix<c_vox[principal+0]; ix+=blockDim.x)
        {
#ifdef SHMCLASS
            voxel[ix] = classify<uchar4>(voxLine[ix]);
#else
            voxel[ix] = voxLine[ix];
#endif
        }
        __syncthreads();
#endif
#endif

#ifndef NOOP
        // Traverse intermediate image pixels which correspond to the current slice.
        // 1 is subtracted from each loop counter to remain inside of the volume boundaries:
#ifdef THREADPERVOXEL
        for (int ix=threadIdx.x; ix<c_vox[principal+0]; ix+=blockDim.x)
#else
        for (int ix=threadIdx.x+from.x; ix<c_vox[principal+0]+iPosX; ix+=blockDim.x)
#endif
        {
#ifndef THREADPERVOXEL
            if(ix<iPosX)
                continue;
#endif
#ifdef THREADPERVOXEL
            const int vidx = ix;
            const int iidx = ix + iPosX;
#else
            const int vidx = ix - iPosX;
            const int iidx = ix;
#endif

            // pointer to destination pixel
            Pixel *pix = imgLine + iidx;
            Pixel d = *pix;
            if(earlyRayTerm && isOpaque(d))
                continue;

#ifdef VOLTEX3D
#if VOLTEX3D == 3
            const float v = tex3D(tex_raw, vidx, c_vox[principal+1]+iPosY-line-1, slice);
#else
            float v;
            switch(principal)
            {
                case 0:
                    v = tex3D(tex_raw, c_vox[2]-slice-1, c_vox[0]-vidx-1, c_vox[1]+iPosY-line-1);
                    break;
                case 1:
                    v = tex3D(tex_raw, line-iPosY, slice, c_vox[1]-vidx-1);
                    break;
                case 2:
                    v = tex3D(tex_raw, vidx, c_vox[0]+iPosY-line-1, slice);
                    break;
            }
#endif
            const float4 c = classify<float4>(v);
#else
#ifdef CONSTDATA
            const float4 c = classify<float4>(uchar(ix));
#else
#ifdef SHMCLASS
            const uchar4 v = *(voxel + BPV * vidx);
            const float4 c = make_float4(v.x/255.f, v.y/255.f, v.z/255.f, v.w/255.f);
#else
            // fetch scalar voxel value
#ifdef SHMLOAD
            const Scalar *v = voxel + BPV * vidx;
#else
            const Scalar *v = voxLine + BPV * vidx;
#endif
            // apply transfer function
            const float4 c = classify<float4>(*v);
#endif
#endif
#endif

            // blend
            const float w = d.w*c.w;
            d.x += w*c.x;
            d.y += w*c.y;
            d.z += w*c.z;
            d.w -= w;

            // store into shmem
            *pix = d;
#ifdef THREADPERVOXEL
            __syncthreads();
#endif
        }
#endif
    }

#ifndef NODISPLAY
    // copy line to intermediate image
    for (int ix=threadIdx.x+from.x; ix<to.x; ix+=blockDim.x)
    {
        uchar4 *dest = img + line*width+ix;
        blend(dest, imgLine[ix]);
    }
#endif
}


template<typename Pixel, int principal, int sliceStep, bool preInt>
struct Ray
{
};

template<typename Pixel, int principal, int sliceStep>
struct Ray<Pixel, principal, sliceStep, false>
{
    Pixel d;

    __device__ Ray()
    {
        initPixel(&d);
    }

    __device__ void accumulate(float x, float y, float z)
    {
        const float v = volume(x, y, z, principal);
        const float4 c = classify<float4>(v);

        // blend
        const float w = d.w*c.w;
        d.x += w*c.x;
        d.y += w*c.y;
        d.z += w*c.z;
        d.w -= w;
    }
};

template<typename Pixel, int principal, int sliceStep>
struct Ray<Pixel, principal, sliceStep, true>
{
    Pixel d;
    float sf;

    __device__ Ray()
        : sf(-1.f)
    {
        initPixel(&d);
    }

    __device__ void accumulate(float x, float y, float z)
    {
        const float sb = volume(x, y, z, principal);
        if(sf >= 0.f)
        {
            const float4 c = tex2D(tex_preint, sf, sb);
            // blend
            const float w = d.w*c.w;
            d.x += w*c.x;
            d.y += w*c.y;
            d.z += w*c.z;
            d.w -= w;
        }
        sf = sb;
    }
};

__device__ bool outsideBounds(int2 p, int2 from1, int2 to1, int2 from2, int2 to2)
{
    return (p.x < from1.x && p.x < from2.x)
        || (p.y < from1.y && p.y < from2.y)
        || (p.x >= to1.x && p.x >= to2.x)
        || (p.y >= to1.y && p.y >= to2.y);
}

__device__ bool fullyInsideIntersection(int2 p1, int2 p2, int2 from1, int2 to1, int2 from2, int2 to2)
{
    return p1.x >= from1.x && p1.x >= from2.x
        && p1.y >= from1.y && p1.y >= from2.y
        && p2.x < to1.x && p2.x < to2.x
        && p2.y < to1.y && p2.y < to2.y;
}

#ifdef VOLTEX3D
template<typename Scalar, int BPV, typename Pixel, int sliceStep, int principal, bool earlyRayTerm, bool preInt>
__global__ void compositeSlicesBilinear(
      uchar4 * __restrict__ img, int width, int height,
#ifdef PITCHED
      const hipPitchedPtr pvoxels,
#else
      const Scalar * __restrict__ voxels,
#endif
      int firstSlice, int lastSlice,
      int2 from, int2 to, int nslice, float scale)
{
#ifdef PATCHES
    Ray<Pixel, principal, sliceStep, preInt> ray;
    int2 p(coord(from));

    float2 tc = make_float2((p.x-c_start[firstSlice].x)*c_tcStep[firstSlice].x+c_tcStart[firstSlice].x,
            (p.y-c_start[firstSlice].y)*c_tcStep[firstSlice].y+c_tcStart[firstSlice].y);
    float2 tc_inc = make_float2((p.x-c_start[firstSlice+sliceStep].x)*c_tcStep[firstSlice+sliceStep].x+c_tcStart[firstSlice+sliceStep].x,
            (p.y-c_start[firstSlice+sliceStep].y)*c_tcStep[firstSlice+sliceStep].y+c_tcStart[firstSlice+sliceStep].y);
    tc_inc.x -= tc.x;
    tc_inc.y -= tc.y;

    // composite slices for this image line
    for (int sliceb=firstSlice; sliceStep>0 ? sliceb<lastSlice : sliceb>lastSlice; sliceb += sliceStep*SliceStack)
    {
        if(earlyRayTerm && isOpaque(ray.d))
            break;

        int last = lastSlice;
        if(sliceStep > 0 && last>sliceb+SliceStack)
            last = sliceb+SliceStack;
        if(sliceStep < 0 && last<sliceb-SliceStack)
            last = sliceb-SliceStack;

        if(outsideBounds(p, c_start[sliceb], c_stop[sliceb],
                    c_start[last-sliceStep], c_stop[last-sliceStep]))
        {
            tc.x += SliceStack*tc_inc.x;
            tc.y += SliceStack*tc_inc.y;
            continue;
        }

#ifdef UNROLL
        const int2 p1 = make_int2(blockDim.x*blockIdx.x+from.x,
                blockDim.y*blockIdx.y+from.y);
        const int2 p2 = make_int2(blockDim.x*blockIdx.x+from.x+blockDim.x-1,
                blockDim.y*blockIdx.y+from.y+blockDim.y-1);
        if(fullyInsideIntersection(p1, p2,
                    c_start[sliceb], c_stop[sliceb],
                    c_start[last-sliceStep], c_stop[last-sliceStep]))
        {
#pragma unroll 4
            for (int slice=sliceb; sliceStep>0 ? slice<last : slice>last; slice += sliceStep)
            {
                ray.accumulate(tc.x, tc.y, c_tc3[slice]);
                tc.x += tc_inc.x;
                tc.y += tc_inc.y;
            }
        }
        else
#endif
        {
            for (int slice=sliceb; sliceStep>0 ? slice<last : slice>last; slice += sliceStep)
            {
                if(earlyRayTerm && isOpaque(ray.d))
                    break;

                if(p.y < c_start[slice].y
                        || p.y >= c_stop[slice].y
                        || p.x<c_start[slice].x
                        || p.x>=c_stop[slice].x)
                {
                    tc.x += tc_inc.x;
                    tc.y += tc_inc.y;
                    continue;
                }

                ray.accumulate(tc.x, tc.y, c_tc3[slice]);
                tc.x += tc_inc.x;
                tc.y += tc_inc.y;
            }
        }
    }

    // copy pixel to intermediate image
    if(p.x >= from.x && p.x < to.x && p.y >= from.y && p.y < to.y)
    {
        uchar4 *dest = img + p.y*width+p.x;
        blend(dest, ray.d);
    }
#else
    // this block's line from the intermediate image
    const int line = blockIdx.x+from.y;
    if (line >= to.y)
        return;

#ifdef NOSHMEM
    for (int ix=threadIdx.x+from.x; ix<to.x; ix+=blockDim.x)
    {
        Ray<Pixel, principal, preInt> ray;

        // composite slices for this image line
        for (int slice=firstSlice; slice!=lastSlice; slice += sliceStep)
        {
            if(earlyRayTerm && isOpaque(ray.d))
                break;

            // compute upper left image corner
            const int iPosY = c_start[slice].y;

            const int iPosX = c_start[slice].x;
            const int endX = c_stop[slice].x;

            if(line < iPosY)
                continue;
            if(line >= c_stop[slice].y)
                continue;

            // Traverse intermediate image pixels which correspond to the current slice.
            // 1 is subtracted from each loop counter to remain inside of the volume boundaries:
            if(ix<iPosX)
                continue;
            if(ix>=endX)
                continue;

            const int vidx = ix - iPosX;
            ray.accumulate(vidx, line-iPosY, slice);
        }

        // copy pixel to intermediate image
        uchar4 *dest = img + line*width+ix;
        blend(dest, ray.d);
    }
#else
    // initialise intermediate image line
    extern __shared__ char smem[];
    Pixel *imgLine = (Pixel *)smem;
    float *sf = (float *)&imgLine[width*preInt];

    for (int ix=threadIdx.x+from.x; ix<to.x; ix+=blockDim.x)
    {
        initPixel(&imgLine[ix]);
        if(preInt)
            sf[ix] = -1.f;
    }

    // composite slices for this image line
    for (int slice=firstSlice; slice!=lastSlice; slice += sliceStep)
    {
        // compute upper left image corner
        const int iPosY = c_start[slice].y;

        if(line < iPosY)
            continue;
        if(line >= c_stop[slice].y)
            continue;

        const int iPosX = c_start[slice].x;
        const int endX = c_stop[slice].x;

        // Traverse intermediate image pixels which correspond to the current slice.
        // 1 is subtracted from each loop counter to remain inside of the volume boundaries:
        for (int ix=threadIdx.x+from.x; ix<endX; ix+=blockDim.x)
        {
            if(ix<iPosX)
                continue;
            const int vidx = ix - iPosX;
            const int iidx = ix;

            // pointer to destination pixel
            Pixel *pix = imgLine + iidx;
            Pixel d = *pix;
            if(earlyRayTerm && isOpaque(d))
                continue;

            if(preInt)
            {
                const float sb = volume(vidx, line-iPosY, slice, principal);
                if(sf[ix] >= 0.f)
                {
                    const float4 c = tex2D(tex_preint, sf[ix], sb);

                    // blend
                    const float w = d.w*c.w;
                    d.x += w*c.x;
                    d.y += w*c.y;
                    d.z += w*c.z;
                    d.w -= w;

                    // store into shmem
                    *pix = d;
                }
                sf[ix] = sb;
            }
            else
            {
                const float v = volume(vidx, line-iPosY, slice, principal);
                const float4 c = classify<float4>(v);

                // blend
                const float w = d.w*c.w;
                d.x += w*c.x;
                d.y += w*c.y;
                d.z += w*c.z;
                d.w -= w;

                // store into shmem
                *pix = d;
            }
        }
    }

    // copy line to intermediate image
    for (int ix=threadIdx.x+from.x; ix<to.x; ix+=blockDim.x)
    {
        uchar4 *dest = img + line*width+ix;
        blend(dest, imgLine[ix]);
    }
#endif
#endif
}

template<typename Scalar, int BPV, typename Pixel, int sliceStep, int principal, bool earlyRayTerm, bool preInt>
__global__ void compositeSORC(
      uchar4 * __restrict__ img, int width, int height,
#ifdef PITCHED
      const hipPitchedPtr pvoxels,
#else
      const Scalar * __restrict__ voxels,
#endif
      int firstSlice, int lastSlice,
      int2 from, int2 to, int nslice, float scale)
{
    Ray<Pixel, principal, sliceStep, preInt> ray;
    int2 p(coord(from));

    float2 tc = make_float2((p.x-c_start[firstSlice].x)*c_tcStep[firstSlice].x+c_tcStart[firstSlice].x,
            (p.y-c_start[firstSlice].y)*c_tcStep[firstSlice].y+c_tcStart[firstSlice].y);
    float2 tc_inc = make_float2((p.x-c_start[firstSlice+sliceStep].x)*c_tcStep[firstSlice+sliceStep].x+c_tcStart[firstSlice+sliceStep].x,
            (p.y-c_start[firstSlice+sliceStep].y)*c_tcStep[firstSlice+sliceStep].y+c_tcStart[firstSlice+sliceStep].y);
    float z = sliceStep == -1 ? 1.f : 0.f;
#if 1
    if(principal == 0)
    {
        z = sliceStep == -1 ? 0.f : 1.f;
    }
#endif
    tc_inc.x -= tc.x;
    tc_inc.y -= tc.y;
    tc_inc.x *= scale;
    tc_inc.y *= scale;

    // composite slices for this image line
    for(int sl =0; sl<nslice; ++sl)
    {
        if(earlyRayTerm && isOpaque(ray.d))
            break;;

        if(tc.x >= 0.f && tc.x < 1.f
                && tc.y >= 0.f && tc.y < 1.f)
        {
#if 0
            float2 mm = minmax(tc.x, tc.y, z, principal);
            float op = tex2D(tex_minmaxTable, mm.x, mm.y);
            if(op == 0.f)
            {
                tc.x += 16*tc_inc.x;
                tc.y += 16*tc_inc.y;
                if(principal == 0)
                    z += 16*c_zStep;
                else
                    z -= 16*c_zStep;
            }
#endif
                ray.accumulate(tc.x, tc.y, z);
        }
        tc.x += tc_inc.x;
        tc.y += tc_inc.y;
#if 1
        if(principal == 0)
            z += c_zStep;
        else
#endif
            z -= c_zStep;
        if(sliceStep==1 && z > 1.f)
            break;
        if(sliceStep==-1 && z < 0.f)
            break;
    }

    // copy pixel to intermediate image
    if(p.x >= from.x && p.x < to.x && p.y >= from.y && p.y < to.y)
    {
        uchar4 *dest = img + p.y*width+p.x;
        blend(dest, ray.d);
    }
}
#endif

//----------------------------------------------------------------------------
// host code
//----------------------------------------------------------------------------


//----------------------------------------------------------------------------
/** Constructor.
  @param vd volume description of volume to display
  @see vvRenderer
*/
template<class Base>
vvCudaSW<Base>::vvCudaSW(vvVolDesc* vd, vvRenderState rs) : Base(vd, rs)
{
   vvDebugMsg::msg(1, "vvCudaSW::vvCudaSW()");

   if(Base::rendererType == Base::SOFTPAR)
       Base::rendererType = Base::CUDAPAR;
   else if(Base::rendererType == Base::SOFTPER)
       Base::rendererType = Base::CUDAPER;

   bool ok = true;

   interSliceInt = false;

   uchar *minArr = NULL, *maxArr = NULL;
   if(Base::vd->bpc == 1)
   {
       const int ds = 16; // downsampling factor
       int vox[3];
       for(int i=0; i<3; ++i)
           vox[i] = (Base::vd->vox[i]+ds-1)/ds;

       minArr = new uchar[vox[0]*vox[1]*vox[2]];
       maxArr = new uchar[vox[0]*vox[1]*vox[2]];

       vd->computeMinMaxArrays(minArr, maxArr, ds);
       hipExtent extent = make_hipExtent(vox[0], vox[1], vox[2]);
       hipChannelFormatDesc desc = hipCreateChannelDesc<uchar>();
       vvCuda::checkError(&ok,
               hipMalloc3DArray(&d_minarr, &desc, extent, 0), "hipMalloc3DArray min");
       vvCuda::checkError(&ok,
               hipMalloc3DArray(&d_maxarr, &desc, extent, 0), "hipMalloc3DArray max");
       hipMemcpy3DParms parms = {0};
       parms.kind = hipMemcpyHostToDevice;
       parms.extent = make_hipExtent(vox[0], vox[1], vox[2]);
       parms.srcPtr = make_hipPitchedPtr(minArr, vox[0], vox[0], vox[1]);
       parms.dstArray = d_minarr;
       vvCuda::checkError(&ok, hipMemcpy3D(&parms), "hipMemcpy3D min");
       parms.srcPtr = make_hipPitchedPtr(maxArr, vox[0], vox[0], vox[1]);
       parms.dstArray = d_maxarr;
       vvCuda::checkError(&ok, hipMemcpy3D(&parms), "hipMemcpy3D max");

       delete[] minArr;
       delete[] maxArr;
       minArr = NULL;
       maxArr = NULL;
   }

   oldLutDist = -1.f;
   imagePrecision = 8;
   earlyRayTerm = true;

   delete Base::intImg; // already allocated as vvSoftImg by vvSoftPer/vvSoftPar
   // we need a power-of-2 image size for glTexImage2D
   int imgSize = vvToolshed::getTextureSize(2 * ts_max(vd->vox[0], vd->vox[1], vd->vox[2]));
   Base::intImg = new vvCudaImg(imgSize, imgSize);
   if(static_cast<vvCudaImg*>(Base::intImg)->getMode() == vvCudaImg::TEXTURE)
       Base::setWarpMode(Base::CUDATEXTURE);

   setQuality(Base::_renderState._quality);

#ifdef FLOATDATA
   for(int i=0; i<3; ++i)
   {
       size_t vox = vd->vox[0]*vd->vox[1]*vd->vox[2];
       fraw[i] = new float[vox];
       for(size_t j=0; j<vox; ++j)
       {
           fraw[i][j] = raw[i][j] / 255.f;
       }
   }
#endif

#if defined(PITCHED) || defined(VOLTEX3D)
#ifdef VOLTEX3D
   for (int i=0; i<VOLTEX3D; ++i)
#else
   for (int i=0; i<3; ++i)
#endif
   {
#ifdef PITCHED
       hipExtent extent = make_hipExtent(vd->vox[(i+1)%3]*sizeof(Scalar), vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
       if(!vvCuda::checkError(&ok, hipMalloc3D(&d_voxptr[i], extent), "hipMalloc3D vox"))
           break;
#else
#if defined(VOLTEX3D) && VOLTEX3D == 1
       hipExtent extent = make_hipExtent(vd->vox[0], vd->vox[1], vd->vox[2]);
#else
       hipExtent extent = make_hipExtent(vd->vox[(i+1)%3], vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
#endif
       hipChannelFormatDesc desc = hipCreateChannelDesc<Scalar>();
       if(!vvCuda::checkError(&ok, hipMalloc3DArray(&d_voxarr[i], &desc, extent, 0), "hipMalloc3DArray vox"))
           break;
#endif
       hipMemcpy3DParms parms = {0};
#if defined(VOLTEX3D) && VOLTEX3D == 1
#ifdef FLOATDATA
       parms.srcPtr = make_hipPitchedPtr(fraw[2], sizeof(Scalar)*vd->vox[0], vd->vox[0], vd->vox[1]);
#else
       parms.srcPtr = make_hipPitchedPtr(vd->getRaw(), sizeof(Scalar)*vd->vox[0], vd->vox[0], vd->vox[1]);
#endif
#else
#ifdef FLOATDATA
       parms.srcPtr = make_hipPitchedPtr(fraw[i], sizeof(Scalar)*vd->vox[(i+1)%3], vd->vox[(i+1)%3], vd->vox[(i+2)%3]);
#else
       parms.srcPtr = make_hipPitchedPtr(Base::raw[i], sizeof(Scalar)*vd->vox[(i+1)%3], vd->vox[(i+1)%3], vd->vox[(i+2)%3]);
#endif
#endif

#ifdef PITCHED
       parms.dstPtr = d_voxptr[i];
       parms.extent = make_hipExtent(vd->vox[(i+1)%3]*sizeof(Scalar), vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
#else
       parms.dstArray = d_voxarr[i];
#if defined(VOLTEX3D) && VOLTEX3D == 1
       parms.extent = make_hipExtent(vd->vox[0], vd->vox[1], vd->vox[2]);
#else
       parms.extent = make_hipExtent(vd->vox[(i+1)%3], vd->vox[(i+2)%3], vd->vox[(i+3)%3]);
#endif
#endif
       parms.kind = hipMemcpyHostToDevice;
       if(!vvCuda::checkError(&ok, hipMemcpy3D(&parms), "hipMemcpy3D vox"))
           break;
   }
#else
   // alloc memory for voxel arrays (for each principal viewing direction)
   vvCuda::checkError(&ok, hipMalloc(&d_voxels, sizeof(Scalar)*vd->vox[0]*vd->vox[1]*vd->vox[2]*3), "hipMalloc vox");
   for (int i=0; i<3; ++i)
   {
#ifdef FLOATDATA
       if (!vvCuda::checkError(&ok, hipMemcpy(d_voxels+i*sizeof(Scalar)*vd->vox[0]*vd->vox[1]*vd->vox[2],
                   fraw[i], sizeof(Scalar)*vd->getFrameBytes(), hipMemcpyHostToDevice), "hipMemcpy vox"))
#else
       if (!vvCuda::checkError(&ok, hipMemcpy(d_voxels+i*sizeof(Scalar)*vd->vox[0]*vd->vox[1]*vd->vox[2],
                   raw[i], vd->getFrameBytes(), hipMemcpyHostToDevice), "hipMemcpy vox"))
#endif
          break;
   }
#endif

   // transfer function is stored as a texture
   vvCuda::checkError(&ok, hipMalloc(&d_tf, Base::getLUTSize()*sizeof(LutEntry)), "hipMalloc tf");
   vvCuda::checkError(&ok, hipBindTexture(NULL, tex_tf, d_tf, Base::getLUTSize()*sizeof(LutEntry)), "bind tf tex");

   // pre-integration table
   hipChannelFormatDesc desc = hipCreateChannelDesc<LutEntry>();
   vvCuda::checkError(&ok, hipMallocArray(&d_preint, &desc, Base::PRE_INT_TABLE_SIZE, Base::PRE_INT_TABLE_SIZE), "hipMalloc preint");
   tex_preint.normalized = true;
   tex_preint.filterMode = Base::bilinLookup ? hipFilterModeLinear : hipFilterModePoint;
   tex_preint.addressMode[0] = hipAddressModeClamp;
   tex_preint.addressMode[1] = hipAddressModeClamp;
   vvCuda::checkError(&ok, hipBindTextureToArray(tex_preint, d_preint, desc), "bind preint tex");

   // min-max-table
   hipChannelFormatDesc descMinMaxTable = hipCreateChannelDesc<uchar>();
   vvCuda::checkError(&ok, hipMallocArray(&d_minmaxTable, &descMinMaxTable, Base::getLUTSize(), Base::getLUTSize()), "hipMalloc minmax");
   tex_minmaxTable.normalized = true;
   tex_minmaxTable.filterMode = hipFilterModePoint;
   tex_minmaxTable.addressMode[0] = hipAddressModeClamp;
   tex_minmaxTable.addressMode[1] = hipAddressModeClamp;
   vvCuda::checkError(&ok, hipBindTextureToArray(tex_minmaxTable, d_minmaxTable, descMinMaxTable), "bind minmax tex");

   // copy volume size (in voxels)
   int h_vox[5];
   for (int i=0; i<5; ++i)
       h_vox[i] = vd->vox[(i+1)%3];
   vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_vox), h_vox, sizeof(int)*5), "hipMemcpy vox");

   Base::updateTransferFunction();
}


//----------------------------------------------------------------------------
/// Destructor.
template<class Base>
vvCudaSW<Base>::~vvCudaSW()
{
   vvDebugMsg::msg(1, "vvCudaSW::~vvCudaSW()");

#ifdef FLOATDATA
   for(int i=0; i<3; ++i)
     delete[] fraw[i];
#endif

   hipUnbindTexture(tex_raw);
   hipUnbindTexture(tex_tf);
   hipFree(d_tf);

   hipUnbindTexture(tex_preint);
   hipFree(d_preint);
#ifdef VOLTEX3D
   for(int i=0; i<VOLTEX3D; ++i)
     hipFree(d_voxarr[i]);
#else
#ifdef PITCHED
   for(int i=0; i<3; ++i)
       hipFree(d_voxptr[i].ptr);
#else
   hipFree(d_voxels);
#endif
#endif
}

template<class Base>
void vvCudaSW<Base>::findAxisRepresentations()
{
#if !defined(VOLTEX3D) || VOLTEX3D!=1
    Base::findAxisRepresentations();
#endif
}

template<class Base>
void vvCudaSW<Base>::updateLUT(float dist)
{
    vvDebugMsg::msg(3, "vvCudaSW::updateLUT()", dist);

    float corr[4];                                  // gamma/alpha corrected RGBA values [0..1]

    const int lutEntries = Base::getLUTSize();
    for (int i=0; i<lutEntries; ++i)
    {
        // Gamma correction:
        if (Base::_renderState._gammaCorrection)
        {
            corr[0] = gammaCorrect(Base::rgbaTF[i * 4],     Base::VV_RED);
            corr[1] = gammaCorrect(Base::rgbaTF[i * 4 + 1], Base::VV_GREEN);
            corr[2] = gammaCorrect(Base::rgbaTF[i * 4 + 2], Base::VV_BLUE);
            corr[3] = gammaCorrect(Base::rgbaTF[i * 4 + 3], Base::VV_ALPHA);
        }
        else
        {
            corr[0] = Base::rgbaTF[i * 4];
            corr[1] = Base::rgbaTF[i * 4 + 1];
            corr[2] = Base::rgbaTF[i * 4 + 2];
            corr[3] = Base::rgbaTF[i * 4 + 3];
        }

        // Opacity correction:
        // for 0 distance draw opaque slices
        if (dist<=0.0 || (Base::_renderState._clipMode && Base::_renderState._clipOpaque)) corr[3] = 1.0f;
        else if (Base::opCorr) corr[3] = 1.0f - powf(1.0f - corr[3], dist);

        // Convert float to uchar and copy to rgbaLUT array:
        for (int c=0; c<4; ++c)
        {
            Base::rgbaConv[i][c] = uchar(corr[c] * 255.0f);
        }
    }

    // update min-max-table
    uchar *minmax = new uchar[lutEntries*lutEntries];
    Base::vd->tf.makeMinMaxTable(lutEntries, minmax);
    vvCuda::checkError(NULL, hipMemcpyToArray(d_minmaxTable, 0, 0, minmax,
                lutEntries*lutEntries, hipMemcpyHostToDevice), "hipMemcpy minmax");
    delete[] minmax;

    // Make pre-integrated LUT:
    if (Base::preIntegration)
    {
        //Base::makeLookupTextureOptimized(dist);           // use this line for fast pre-integration LUT
        Base::makeLookupTextureCorrect(dist);   // use this line for slow but more correct pre-integration LUT
    }

    vvCuda::checkError(NULL, hipMemcpy(d_tf, Base::rgbaConv, Base::getLUTSize()*sizeof(LutEntry), hipMemcpyHostToDevice), "hipMemcpy tf");
    if(Base::preIntegration)
    {
        vvCuda::checkError(NULL, hipMemcpyToArray(d_preint, 0, 0, &Base::preIntTable[0][0][0],
                    Base::PRE_INT_TABLE_SIZE*Base::PRE_INT_TABLE_SIZE*sizeof(LutEntry), hipMemcpyHostToDevice), "hipMemcpy preint");
    }
}

template<class Base, typename Pixel, int principal, int sliceStep, bool earlyRayTerm>
CompositionFunction selectComposition(vvCudaSW<Base> *rend)
{
#ifdef VOLTEX3D
    if(rend->getSliceInterpol() || rend->getRendererType() == vvRenderer::CUDAPER)
    {
        if(rend->getInterSliceInterpol() || 1)
        {
            if(rend->getPreIntegration())
                return compositeSORC<Scalar, 1, Pixel, sliceStep, principal, earlyRayTerm, true>;
            else
                return compositeSORC<Scalar, 1, Pixel, sliceStep, principal, earlyRayTerm, false>;
        }
        else
        {
            if(rend->getPreIntegration())
                return compositeSlicesBilinear<Scalar, 1, Pixel, sliceStep, principal, earlyRayTerm, true>;
            else
                return compositeSlicesBilinear<Scalar, 1, Pixel, sliceStep, principal, earlyRayTerm, false>;
        }
    }
    else
#endif
    {
        if(rend->getRendererType() == vvRenderer::CUDAPAR)
            return compositeSlicesNearest<Scalar, 1, Pixel, sliceStep, principal, earlyRayTerm>;
    }

    return NULL;
}

template<class Base, typename Pixel, int principal, int sliceStep>
CompositionFunction selectCompositionWithEarlyTermination(vvCudaSW<Base> *rend)
{
    if(rend->getEarlyRayTerm())
        return selectComposition<Base, Pixel, principal, sliceStep, true>(rend);
    else
        return selectComposition<Base, Pixel, principal, sliceStep, false>(rend);
}

template<class Base, typename Pixel, int principal>
CompositionFunction selectCompositionWithSliceStep(vvCudaSW<Base> *rend, int sliceStep)
{
    switch(sliceStep)
    {
        case 1:
            return selectCompositionWithEarlyTermination<Base, Pixel, principal,1>(rend);
        case -1:
            return selectCompositionWithEarlyTermination<Base, Pixel, principal,-1>(rend);
        default:
            assert("slice step out of range" == NULL);
    }

    return NULL;
}

template<class Base, typename Pixel>
CompositionFunction selectCompositionWithPrincipal(vvCudaSW<Base> *rend, int sliceStep)
{
    switch(rend->getPrincipal())
    {
        case 0:
            return selectCompositionWithSliceStep<Base, Pixel, 0>(rend, sliceStep);
        case 1:
            return selectCompositionWithSliceStep<Base, Pixel, 1>(rend, sliceStep);
        case 2:
            return selectCompositionWithSliceStep<Base, Pixel, 2>(rend, sliceStep);
        default:
            assert("principal axis out of range" == NULL);

    }

    return NULL;
}

template<class Base>
CompositionFunction selectCompositionWithPrecision(vvCudaSW<Base> *rend, int sliceStep)
{
    switch(rend->getPrecision())
    {
        case 8:
            return selectCompositionWithPrincipal<Base, uchar4>(rend, sliceStep);
        case 32:
            return selectCompositionWithPrincipal<Base, float4>(rend, sliceStep);
        default:
            assert("invalid precision" == NULL);
    }

    return NULL;
}

//----------------------------------------------------------------------------
/** Composite the volume slices to the intermediate image.
  The function prepareRendering() must be called before this method.
  The shear transformation matrices have to be computed before calling this method.
  The volume slices are processed from front to back.
  @param from,to optional arguments to define first and last intermediate image line to render.
                 if not passed, the entire intermediate image will be rendered
*/
template<class Base>
void vvCudaSW<Base>::compositeVolume(int fromY, int toY)
{
   vvDebugMsg::msg(3, "vvCudaSW::compositeVolume(): ", fromY, toY);

   // If stacking==true then draw front to back, else draw back to front:
   int firstSlice = (Base::stacking) ? 0 : (Base::len[2]-1);  // first slice to process
   int lastSlice  = (Base::stacking) ? (Base::len[2]-1) : 0;  // last slice to process
   int sliceStep  = (Base::stacking) ? 1 : -1;          // step size to get to next slice

   Base::earlyRayTermination = 0;

   if (fromY == -1)
       fromY = 0;
   if (toY == -1)
       toY = Base::intImg->height;

   // compute data for determining upper left image corner of each slice and copy it to device
   vvVector4 start, end;
   Base::findSlicePosition(firstSlice, &start, &end);
   vvVector4 sinc, einc;
   Base::findSlicePosition(firstSlice+sliceStep, &sinc, &einc);
   sinc.sub(&start);
   einc.sub(&end);

   float dist = sqrtf(1.0f + sinc.e[0] * sinc.e[0] + sinc.e[1] * sinc.e[1]);

   float q = Base::_renderState._quality;
   float s = 1.f/dist;
   int nslice = Base::len[2]/s;
#if 0
   for(int i=0; i<4; ++i)
   {
       sinc.e[i] *= s;
       einc.e[i] *= s;
   }
#endif

   float zstep = -s / Base::vd->vox[Base::principal];
   switch(Base::principal)
   {
       case 0:
           zstep = -(float)sliceStep / Base::vd->vox[Base::principal];
           break;
       case 1:
           zstep = -(float)sliceStep / Base::vd->vox[Base::principal];
           break;
       case 2:
           zstep = -(float)sliceStep / Base::vd->vox[Base::principal];
           break;
   }
   zstep *= s;
   //fprintf(stderr, "nslice=%d, step=%f, tot=%f\n", nslice, zstep, nslice*zstep);
#if 0
   fprintf(stderr, "step=%d, zstep=%f, princ=%d\n",
           sliceStep, zstep, Base::principal);
#endif

   hipMemcpyToSymbol(HIP_SYMBOL(c_zStep), &zstep, sizeof(float));

   dist = 1.f/Base::_renderState._quality;

   if(oldLutDist/dist < 0.9f || dist/oldLutDist < 0.9f)
   {
       updateLUT(dist);
       oldLutDist = dist;
   }

   int2 from = make_int2(0, fromY);
   int2 to = make_int2(Base::intImg->width, toY);

   if(Base::sliceInterpol)
   {
       from = make_int2(Base::intImg->width, Base::intImg->height);
       to = make_int2(0, 0);
   }

   vvVector4 scur = start;
   vvVector4 ecur = end;
#if defined(VOLTEX3D) && VOLTEX3D==1
   const int p = Base::principal;
#else
   const int p = 2;
#endif
   for(int slice=firstSlice; slice != lastSlice; slice += sliceStep)
   {
#ifdef VOLTEX3D
       if(Base::sliceInterpol)
       {
           const float sx = scur.e[0]/scur.e[3];
           const float sy = scur.e[1]/scur.e[3];
           const float ex = ecur.e[0]/ecur.e[3];
           const float ey = ecur.e[1]/ecur.e[3];

           h_start[slice].x = max(0,int(floor(sx)));
           h_start[slice].y = max(0,int(floor(sy)));

           from.x = min(from.x, h_start[slice].x);
           from.y = min(from.y, h_start[slice].y);

           h_stop[slice].x = min(Base::intImg->width-1,int(ceil(ex)));
           h_stop[slice].y = min(Base::intImg->height-1,int(ceil(ey)));

           to.x = max(to.x, h_stop[slice].x);
           to.y = max(to.y, h_stop[slice].y);

           switch(p)
           {
               case 0:
                   h_tcStep[slice].x = -1.f/(ex-sx);
                   h_tcStep[slice].y = -1.f/(ey-sy);

                   h_tcStart[slice].x = 1.f + (h_start[slice].x - sx + 0.5f)*h_tcStep[slice].x;
                   h_tcStart[slice].y = 1.f + (h_start[slice].y - sy + 0.5f)*h_tcStep[slice].y;

                   h_tc3[slice] = 1.f-(slice+0.5f)*1.f/Base::vd->vox[Base::principal];
                   break;
                case 1:
                   h_tcStep[slice].x = -1.f/(ex-sx);
                   h_tcStep[slice].y = 1.f/(ey-sy);

                   h_tcStart[slice].x = 1.f + (h_start[slice].x - sx + 0.5f)*h_tcStep[slice].x;
                   h_tcStart[slice].y = (h_start[slice].y - sy + 0.5f)*h_tcStep[slice].y;

                   h_tc3[slice] = (slice+0.5f)*1.f/Base::vd->vox[Base::principal];
                   break;
                case 2:
                   h_tcStep[slice].x = 1.f/(ex-sx);
                   h_tcStep[slice].y = -1.f/(ey-sy);

                   h_tcStart[slice].x = (h_start[slice].x - sx + 0.5f)*h_tcStep[slice].x;
                   h_tcStart[slice].y = 1.f + (h_start[slice].y - sy + 0.5f)*h_tcStep[slice].y;

                   h_tc3[slice] = (slice+0.5f)*1.f/Base::vd->vox[Base::principal];
                   break;
           }

           ecur.add(&einc);
       }
       else
#endif
       {
           h_start[slice].x = int(scur.e[0] / scur.e[3] + 0.5f);
           h_start[slice].y = int(scur.e[1] / scur.e[3] + 0.5f);
       }
       scur.add(&sinc);
   }
   from.y = max(from.y, fromY);
   to.y = min(to.y, toY);

   //fprintf(stderr, "p=%d: (%d,%d) - (%d,%d)\n", Base::principal, from.x, from.y, to.x, to.y);

   bool ok = true;
   vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_start), h_start, sizeof(h_start)), "hipMemcpy start");
#ifdef VOLTEX3D
   if(Base::sliceInterpol)
   {
       vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_stop), h_stop, sizeof(h_stop)), "hipMemcpy stop");
       vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_tcStart), h_tcStart, sizeof(h_tcStart)), "hipMemcpy tcStart");
       vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_tcStep), h_tcStep, sizeof(h_tcStep)), "hipMemcpy tcStep");
       vvCuda::checkError(&ok, hipMemcpyToSymbol(HIP_SYMBOL(c_tc3), h_tc3, sizeof(h_tc3)), "hipMemcpy tc3");
   }
#endif

#ifdef VOLTEX3D
   tex_raw.normalized = Base::sliceInterpol;
   tex_raw.filterMode = Base::sliceInterpol ? hipFilterModeLinear : hipFilterModePoint;
   tex_raw.addressMode[0] = hipAddressModeClamp;
   tex_raw.addressMode[1] = hipAddressModeClamp;
   tex_raw.addressMode[2] = hipAddressModeClamp;
   hipChannelFormatDesc desc = hipCreateChannelDesc<Scalar>();
#if VOLTEX3D == 1
   hipBindTextureToArray(tex_raw, d_voxarr[0], desc);
#else
   hipBindTextureToArray(tex_raw, d_voxarr[Base::principal], desc);
#endif
#endif

   hipChannelFormatDesc minmaxDesc = hipCreateChannelDesc<uchar>();
   tex_min.normalized = true;
   tex_min.filterMode = hipFilterModePoint;
   tex_min.addressMode[0] = hipAddressModeClamp;
   tex_min.addressMode[1] = hipAddressModeClamp;
   tex_min.addressMode[2] = hipAddressModeClamp;
   hipBindTextureToArray(tex_min, d_minarr, minmaxDesc);
   tex_max.normalized = true;
   tex_max.filterMode = hipFilterModePoint;
   tex_max.addressMode[0] = hipAddressModeClamp;
   tex_max.addressMode[1] = hipAddressModeClamp;
   tex_max.addressMode[2] = hipAddressModeClamp;
   hipBindTextureToArray(tex_max, d_maxarr, minmaxDesc);

   static_cast<vvCudaImg*>(Base::intImg)->map();

   int shmsize = Base::intImg->width*imagePrecision/8*4;
#ifdef SHMLOAD
   shmsize += Base::vd->vox[Base::principal]*Base::vd->getBPV()*sizeof(Scalar);
#endif
   if(Base::preIntegration)
   {
       shmsize += Base::intImg->width*sizeof(float);
   }
#ifdef NOSHMEM
   if(Base::sliceInterpol)
       shmsize = 0;
#endif

   uchar4 *d_img = static_cast<vvCudaImg*>(Base::intImg)->getDImg();
   dim3 grid((Base::intImg->width+Patch.x-1)/Patch.x, (toY-fromY+Patch.y-1)/Patch.y);
   dim3 block = Patch;
   clearImage <<<grid, Patch>>>(d_img, Base::intImg->width, Base::intImg->height, fromY, toY);

   if(CompositionFunction compose = selectCompositionWithPrecision(this, sliceStep))
   {
#ifdef PATCHES
       if(Base::sliceInterpol)
       {
           grid = dim3((to.x-from.x+Patch.x-1)/Patch.x, (to.y-from.y+Patch.y-1)/Patch.y);
       }
       else
#endif
       {
           grid = dim3(to.y-from.y);
           block = dim3(nthreads);
       }

       // do the computation on the device
       for(int i=lastSlice; i*sliceStep>firstSlice*sliceStep; i-=sliceStep*MaxCompositeSlices)
       {
           hipDeviceSynchronize();
#ifdef PITCHED
           compose <<<grid, block, shmsize>>>(
                   d_img, Base::intImg->width, Base::intImg->height,
                   d_voxptr[Base::principal],
                   sliceStep*max(sliceStep*i-MaxCompositeSlices, sliceStep*firstSlice), i,
                   from, to, nslice, s);
#else
           compose <<<grid, block, shmsize>>>(
                   d_img, Base::intImg->width, Base::intImg->height,
                   (Scalar *)(d_voxels+sizeof(Scalar)*Base::vd->getBPV()*Base::principal*(Base::vd->vox[0]*Base::vd->vox[1]*Base::vd->vox[2])),
                   sliceStep*max(sliceStep*i-MaxCompositeSlices, sliceStep*firstSlice), i,
                   from, to, nslice, s);
#endif
       }
   }

#ifdef VOLTEX3D
   hipUnbindTexture(tex_raw);
#endif

   // copy back or unmap for using as PBO
   static_cast<vvCudaImg*>(Base::intImg)->unmap();
}

template<class Base>
void vvCudaSW<Base>::setParameter(typename Base::ParameterType param, float val, char *cval)
{
    vvDebugMsg::msg(3, "vvCudaSW::setParameter()");
    switch(param)
    {
        case Base::VV_IMG_PRECISION:
            if(val == 8)
                imagePrecision = 8;
            else
                imagePrecision = 32;
            break;
        case Base::VV_TERMINATEEARLY:
            earlyRayTerm = (val != 0.f);
            break;
        case Base::VV_INTERSLICEINT:
            interSliceInt = (val != 0.f);
            break;
        default:
            Base::setParameter(param, val, cval);
            break;
    }
}

template<class Base>
float vvCudaSW<Base>::getParameter(typename Base::ParameterType param, char *cval) const
{
    vvDebugMsg::msg(3, "vvCudaSW::getParameter()");
    switch(param)
    {
        case Base::VV_IMG_PRECISION:
            return imagePrecision;
        case Base::VV_TERMINATEEARLY:
            return (earlyRayTerm ? 1.f : 0.f);
        case Base::VV_INTERSLICEINT:
            return (interSliceInt ? 1.f : 0.f);
        default:
            return Base::getParameter(param, cval);
    }
}

//----------------------------------------------------------------------------
/** Set rendering quality.
  When quality changes, the intermediate image must be resized and the shear
  matrix has to be recomputed.
  @see vvRenderer#setQuality
*/
template<>
void vvCudaSW<vvSoftPar>::setQuality(float q)
{
   typedef vvSoftPar Base;

   vvDebugMsg::msg(3, "vvCudaSW::setQuality()", q);

#ifdef VV_XVID
   q = 1.0f;
#endif

   Base::_renderState._quality = q;

   if(!Base::sliceInterpol)
       q = 1.f;

   quality = q;

   // edge size of intermediate image [pixels]
   int intImgSize = (int)((2.0f * q) * ts_max(Base::vd->vox[0], Base::vd->vox[1], Base::vd->vox[2]));
   if (intImgSize<1)
   {
      intImgSize = 1;
      quality = 1.0f / (2.0f * ts_max(Base::vd->vox[0], Base::vd->vox[1], Base::vd->vox[2]));
   }

   intImgSize = ts_clamp(intImgSize, 16, 4096);
   intImgSize = vvToolshed::getTextureSize(intImgSize);

   Base::intImg->setSize(intImgSize, intImgSize);
   vvSoftPar::findShearMatrix();
   vvDebugMsg::msg(3, "Intermediate image edge length: ", intImgSize);
}

template<>
void vvCudaSW<vvSoftPer>::setQuality(float q)
{
   vvDebugMsg::msg(3, "vvCudaSW::setQuality()", q);
   vvSoftPer::setQuality(q);
}

vvCudaPer::vvCudaPer(vvVolDesc *vd, vvRenderState rs)
: vvCudaSW<vvSoftPer>(vd, rs)
{
}

vvCudaPar::vvCudaPar(vvVolDesc *vd, vvRenderState rs)
: vvCudaSW<vvSoftPar>(vd, rs)
{
}
//============================================================================
// End of File
//============================================================================
